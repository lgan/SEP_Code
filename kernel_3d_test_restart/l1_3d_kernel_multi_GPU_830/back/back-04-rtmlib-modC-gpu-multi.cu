#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "gpu.h"

__global__ void rtm_gpu_kernel(int ny, int nz, int nx,
        float *g_ex_Vy0_now,  float * g_ex_Vx0_now, float * g_ex_Vz0_now, float * g_ex_sigmayy0_now, float *g_ex_sigmaxx0_now, float * g_ex_sigmazz0_now, float * g_ex_sigmaxy0_now, float * g_ex_sigmaxz0_now, float * g_ex_sigmayz0_now,//(nz, nx, nt)
        float *g_ex_Vy0_next,  float * g_ex_Vx0_next, float * g_ex_Vz0_next, float * g_ex_sigmayy0_next, float *g_ex_sigmaxx0_next, float * g_ex_sigmazz0_next, float * g_ex_sigmaxy0_next, float * g_ex_sigmaxz0_next, float * g_ex_sigmayz0_next,//(nz, nx, nt)
        float *g_ex_Vy0_pre,  float * g_ex_Vx0_pre, float * g_ex_Vz0_pre, float * g_ex_sigmayy0_pre, float *g_ex_sigmaxx0_pre, float * g_ex_sigmazz0_pre, float * g_ex_sigmaxy0_pre, float * g_ex_sigmaxz0_pre, float * g_ex_sigmayz0_pre,//(nz, nx, nt)
     	float * g_ex_m1_y,    float * g_ex_m1_x,    float * g_ex_m1_z,   float *  g_ex_m2,  float * g_ex_m3,  float * g_ex_m2m3);//(nz+10,	nx+10)




extern "C" void setup_cuda(int n_gpus){
	
fprintf(stderr, "Today, we are using %d GPUs; specifically: \n", n_gpus);
	int dr;
	int i, j, k;

	for(i=0; i<n_gpus; i++) device[i] = i;

	for(i=0; i<n_gpus; i++) {
		hipDeviceSynchronize();
		
		hipSetDevice(device[i]);
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, device[i]);
		hipDriverGetVersion(&dr);

		fprintf(stderr, "   GPU %s (%d),", properties.name, device[i]);
		if(properties.unifiedAddressing) fprintf(stderr, " UVA initiated, ");
		else fprintf(stderr, " UVA not working,");
		fprintf(stderr, "driver %d\n", dr);

		//Enable P2P memcopies between GPUs
		if(n_gpus >1){
			for(j=0;j<n_gpus; j++){
				if(i == j) continue;
				int peer_access_available = 0;
				hipDeviceCanAccessPeer(&peer_access_available, device[i], device[j]);
				if(peer_access_available)
					hipDeviceEnablePeerAccess(device[j], 0);
			}
		}
	}

}


#ifndef MULTI_GPUS
extern "C" void rtm_gpu_init(int ny, int nz, int nx, int n_gpus) 
{	
	hipError_t cuda_ret;
     	hipError_t err;

	//Set Device 
    	cuda_ret = hipSetDevice(1);
	if(cuda_ret != hipSuccess){
		fprintf(stderr, "Failed to Set The cuda Device !\n");
		exit(0);
	}
	else{
		fprintf(stderr, "GPU Device Set ====> OK\n");
	}

	// data init
	
	//Time step +1
	hipMalloc(&g_ex_Vx0_now, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_Vz0_now, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_Vy0_now, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmaxx0_now, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmazz0_now, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmayy0_now, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmaxy0_now, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmaxz0_now, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmayz0_now, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	
	//Time step +2
	hipMalloc(&g_ex_Vx0_next, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_Vz0_next, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_Vy0_next, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmaxx0_next, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmazz0_next, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmayy0_next, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmaxy0_next, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmaxz0_next, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmayz0_next, sizeof(float)*(ny+10)*(nx+10)*(nz+10));


	//time step 0 and output
	hipMalloc(&g_ex_Vx0_pre, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_Vz0_pre, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_Vy0_pre, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmaxx0_pre, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmazz0_pre, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmayy0_pre, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmaxy0_pre, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmaxz0_pre, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmayz0_pre, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
   
	//expaned arrays to store different Operators 
	hipMalloc(&g_ex_m2, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_m3, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_m2m3, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_m1_x, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_m1_y, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_m1_z, sizeof(float)*(ny+10)*(nx+10)*(nz+10));

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(rtm_gpu_kernel),hipFuncCachePreferShared);
	
	err = hipGetLastError();
	if(hipSuccess != err){
		fprintf(stderr, "Cuda error0: %s.\n", hipGetErrorString(err));
		exit(0);
	}else{	
		fprintf(stderr,"GPU Data Init ====> OK\n");
	}
	// data copy

}



extern "C" void rtm_gpu_copy_in(int ny, int nz, int nx, 
        float *ex_Vy0_now,  float * ex_Vx0_now, float * ex_Vz0_now, float * ex_sigmayy0_now, float *ex_sigmaxx0_now, float * ex_sigmazz0_now, float * ex_sigmaxy0_now, float * ex_sigmaxz0_now, float * ex_sigmayz0_now,//(nz, nx, nt)
        float *ex_Vy0_next,  float * ex_Vx0_next, float * ex_Vz0_next, float * ex_sigmayy0_next, float *ex_sigmaxx0_next, float * ex_sigmazz0_next, float * ex_sigmaxy0_next, float * ex_sigmaxz0_next, float * ex_sigmayz0_next,//(nz, nx, nt)
        float *ex_Vy0_pre,  float * ex_Vx0_pre, float * ex_Vz0_pre, float * ex_sigmayy0_pre, float *ex_sigmaxx0_pre, float * ex_sigmazz0_pre, float * ex_sigmaxy0_pre, float * ex_sigmaxz0_pre, float * ex_sigmayz0_pre,//(nz, nx, nt)
     	float * ex_m1_y,  float * ex_m1_x, float * ex_m1_z, float * ex_m2, float * ex_m3, float * ex_m2m3)//(nz+10,	nx+10)
  {	
     	hipError_t err;
	
	// data copy

	hipMemcpy(g_ex_Vy0_now, ex_Vy0_now, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_Vx0_now, ex_Vx0_now, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_Vz0_now, ex_Vz0_now, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmaxx0_now, ex_sigmaxx0_now, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmayy0_now, ex_sigmayy0_now, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmaxy0_now, ex_sigmaxy0_now, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmayz0_now, ex_sigmayz0_now, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmaxz0_now, ex_sigmaxz0_now, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmazz0_now, ex_sigmazz0_now, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);

	hipMemcpy(g_ex_Vy0_next, ex_Vy0_next, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_Vx0_next, ex_Vx0_next, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_Vz0_next, ex_Vz0_next, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmaxx0_next, ex_sigmaxx0_next, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmayy0_next, ex_sigmayy0_next, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmaxy0_next, ex_sigmaxy0_next, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmayz0_next, ex_sigmayz0_next, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmaxz0_next, ex_sigmaxz0_next, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmazz0_next, ex_sigmazz0_next, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);

	hipMemcpy(g_ex_Vy0_pre, ex_Vy0_pre, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_Vx0_pre, ex_Vx0_pre, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_Vz0_pre, ex_Vz0_pre, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmaxx0_pre, ex_sigmaxx0_pre, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmayy0_pre, ex_sigmayy0_pre, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmaxy0_pre, ex_sigmaxy0_pre, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmayz0_pre, ex_sigmayz0_pre, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmaxz0_pre, ex_sigmaxz0_pre, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmazz0_pre, ex_sigmazz0_pre, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);



	hipMemcpy(g_ex_m1_y, ex_m1_y, sizeof(float)*(ny+10)*(nx+10)*(nz+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_m1_x, ex_m1_x, sizeof(float)*(ny+10)*(nx+10)*(nz+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_m1_z, ex_m1_z, sizeof(float)*(ny+10)*(nx+10)*(nz+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_m2, ex_m2, sizeof(float)*(ny+10)*(nx+10)*(nz+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_m3, ex_m3, sizeof(float)*(ny+10)*(nx+10)*(nz+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_m2m3, ex_m2m3, sizeof(float)*(ny+10)*(nx+10)*(nz+10), hipMemcpyHostToDevice);
	
	err = hipGetLastError();
	if(hipSuccess != err){
		fprintf(stderr, "Cuda error1: %s.\n", hipGetErrorString(err));
		exit(0);
		
	}else{
		fprintf(stderr,"Data Copy To GPU  ====> OK\n");
	}
}



extern "C" void rtm_gpu_copy_out(int ny, int nz, int nx, 
        float *ex_Vy0_pre,  float * ex_Vx0_pre, float * ex_Vz0_pre, float * ex_sigmayy0_pre, float *ex_sigmaxx0_pre, float * ex_sigmazz0_pre, float * ex_sigmaxy0_pre, float * ex_sigmaxz0_pre, float * ex_sigmayz0_pre)//(nz, nx, nt)
{	
     	hipError_t err;
	
	// data copy back from GPU mem
	hipMemcpy(ex_Vy0_pre, g_ex_Vy0_pre, sizeof(float)*(nx+10)*(nz+10)*(ny+10),  		hipMemcpyDeviceToHost);
	hipMemcpy(ex_Vx0_pre, g_ex_Vx0_pre, sizeof(float)*(nx+10)*(nz+10)*(ny+10),  		hipMemcpyDeviceToHost);
	hipMemcpy(ex_Vz0_pre, g_ex_Vz0_pre, sizeof(float)*(nx+10)*(nz+10)*(ny+10), 			hipMemcpyDeviceToHost);
	hipMemcpy(ex_sigmaxx0_pre, g_ex_sigmaxx0_pre, sizeof(float)*(nx+10)*(nz+10)*(ny+10), 	hipMemcpyDeviceToHost);
	hipMemcpy(ex_sigmayy0_pre, g_ex_sigmayy0_pre, sizeof(float)*(nx+10)*(nz+10)*(ny+10), 	hipMemcpyDeviceToHost);
	hipMemcpy(ex_sigmaxy0_pre, g_ex_sigmaxy0_pre, sizeof(float)*(nx+10)*(nz+10)*(ny+10), 	hipMemcpyDeviceToHost);
	hipMemcpy(ex_sigmaxz0_pre, g_ex_sigmaxz0_pre, sizeof(float)*(nx+10)*(nz+10)*(ny+10), 	hipMemcpyDeviceToHost);
	hipMemcpy(ex_sigmayz0_pre, g_ex_sigmayz0_pre, sizeof(float)*(nx+10)*(nz+10)*(ny+10), 	hipMemcpyDeviceToHost);
	hipMemcpy(ex_sigmazz0_pre, g_ex_sigmazz0_pre, sizeof(float)*(nx+10)*(nz+10)*(ny+10), 	hipMemcpyDeviceToHost);
	//hipMemcpy(sigmazz0, g_sigmazz0,  sizeof(float)*nx*nz*nt, 	hipMemcpyDeviceToHost);
	err = hipGetLastError();
	if(hipSuccess != err){
		fprintf(stderr, "Cuda error2: %s.\n", hipGetErrorString(err));
		exit(0);
	}else{
	fprintf(stderr,"Data Copy To CPU ====> OK\n");
	}
}


extern "C" void rtm_gpu_final()
{

	//release GPU memory space

     	hipError_t err;
	
	hipFree(g_ex_Vx0_now);
	hipFree(g_ex_Vz0_now);
	hipFree(g_ex_Vy0_now);
	hipFree(g_ex_sigmaxx0_now);
	hipFree(g_ex_sigmazz0_now);
	hipFree(g_ex_sigmayy0_now);
	hipFree(g_ex_sigmaxy0_now);
	hipFree(g_ex_sigmaxz0_now);
	hipFree(g_ex_sigmayz0_now);
	
	//Time step +2
	hipFree(g_ex_Vx0_next);
	hipFree(g_ex_Vz0_next);
	hipFree(g_ex_Vy0_next);
	hipFree(g_ex_sigmaxx0_next);
	hipFree(g_ex_sigmazz0_next);
	hipFree(g_ex_sigmayy0_next);
	hipFree(g_ex_sigmaxy0_next);
	hipFree(g_ex_sigmaxz0_next);
	hipFree(g_ex_sigmayz0_next);


	//time step 0 and output
	hipFree(g_ex_Vx0_pre);
	hipFree(g_ex_Vz0_pre);
	hipFree(g_ex_Vy0_pre);
	hipFree(g_ex_sigmaxx0_pre);
	hipFree(g_ex_sigmazz0_pre);
	hipFree(g_ex_sigmayy0_pre);
	hipFree(g_ex_sigmaxy0_pre);
	hipFree(g_ex_sigmaxz0_pre);
	hipFree(g_ex_sigmayz0_pre);
   
	//expaned arrays to store different Operators 
	hipFree(g_ex_m2);
	hipFree(g_ex_m3);
	hipFree(g_ex_m2m3);
	hipFree(g_ex_m1_x);
	hipFree(g_ex_m1_y);
	hipFree(g_ex_m1_z);

	err = hipGetLastError();

	if(hipSuccess != err){
		fprintf(stderr, "Cuda error3: %s.\n", hipGetErrorString(err));
		exit(0);
	}else{
	fprintf(stderr,"GPU Mem Released ====> OK\n");
	}
}


void rtm_gpu_change_pointer(int n_gpus){
		
			
		fprintf(stderr, "GPU pointer changed\n");

		__device__ float *g_tmp = NULL;

		
		for(int i=0; i<n_gpus; i++){
		hipSetDevice[i];
		g_tmp = 	g_ex_Vx0_pre[i];
		g_ex_Vx0_pre[i] = 	g_ex_Vx0_now[i];
		g_ex_Vx0_now[i] = 	g_tmp;

		g_tmp = 	g_ex_Vx0_pre[i];
		g_ex_Vx0_pre[i] = 	g_ex_Vx0_next[i];
		g_ex_Vx0_next[i] = 	g_tmp; 

	

//		g_tmp = 	g_ex_Vz0_pre;
//		g_ex_Vz0_pre = 	g_ex_Vz0_now;
//		g_ex_Vz0_now = 	g_tmp;
//
//		g_tmp = 	g_ex_Vz0_pre;
//		g_ex_Vz0_pre = 	g_ex_Vz0_next;
//		g_ex_Vz0_next = 	g_tmp; 
//
//
//	
//		g_tmp = 	g_ex_Vy0_pre;
//		g_ex_Vy0_pre = 	g_ex_Vy0_now;
//		g_ex_Vy0_now = 	g_tmp;
//
//		g_tmp = 	g_ex_Vy0_pre;
//		g_ex_Vy0_pre = 	g_ex_Vy0_next;
//		g_ex_Vy0_next = 	g_tmp; 

	

		g_tmp = 	g_ex_sigmaxx0_pre[i];
		g_ex_sigmaxx0_pre[i] = g_ex_sigmaxx0_now[i];
		g_ex_sigmaxx0_now[i] = g_tmp;

		g_tmp = g_ex_sigmaxx0_pre[i];
		g_ex_sigmaxx0_pre[i] = g_ex_sigmaxx0_next[i];
		g_ex_sigmaxx0_next[i] = g_tmp; 

	
		g_tmp = g_ex_sigmazz0_pre[i];
		g_ex_sigmazz0_pre[i] = g_ex_sigmazz0_now[i];
		g_ex_sigmazz0_now[i] = g_tmp;

		g_tmp = g_ex_sigmazz0_pre[i];
		g_ex_sigmazz0_pre[i] = g_ex_sigmazz0_next[i];
		g_ex_sigmazz0_next[i] = g_tmp; 


	
		g_tmp = g_ex_sigmayy0_pre[i];
		g_ex_sigmayy0_pre[i] = g_ex_sigmayy0_now[i];
		g_ex_sigmayy0_now[i] = g_tmp;

		g_tmp = g_ex_sigmayy0_pre[i];
		g_ex_sigmayy0_pre[i] = g_ex_sigmayy0_next[i];
		g_ex_sigmayy0_next[i] = g_tmp; 


	
		g_tmp = g_ex_sigmaxy0_pre[i];
		g_ex_sigmaxy0_pre[i] = g_ex_sigmaxy0_now[i];
		g_ex_sigmaxy0_now[i] = g_tmp;

		g_tmp = g_ex_sigmaxy0_pre[i];
		g_ex_sigmaxy0_pre[i] = g_ex_sigmaxy0_next[i];
		g_ex_sigmaxy0_next[i] = g_tmp; 



	
		g_tmp = g_ex_sigmaxz0_pre[i];
		g_ex_sigmaxz0_pre[i] = g_ex_sigmaxz0_now[i];
		g_ex_sigmaxz0_now[i] = g_tmp;

		g_tmp = g_ex_sigmaxz0_pre[i];
		g_ex_sigmaxz0_pre[i] = g_ex_sigmaxz0_next[i];
		g_ex_sigmaxz0_next[i] = g_tmp; 


	
		g_tmp = g_ex_sigmayz0_pre[i];
		g_ex_sigmayz0_pre[i] = g_ex_sigmayz0_now[i];
		g_ex_sigmayz0_now[i] = g_tmp;
		
		g_tmp = g_ex_sigmayz0_pre[i];
		g_ex_sigmayz0_pre[i] = g_ex_sigmayz0_next[i];
		g_ex_sigmayz0_next[i] = g_tmp; 
		}

}

#endif

__global__ void rtm_gpu_kernel(int ny, int nz, int nx,
        float *g_ex_Vy0_now,  float * g_ex_Vx0_now, float * g_ex_Vz0_now, float * g_ex_sigmayy0_now, float *g_ex_sigmaxx0_now, float * g_ex_sigmazz0_now, float * g_ex_sigmaxy0_now, float * g_ex_sigmaxz0_now, float * g_ex_sigmayz0_now,//(ny+10, nx+10, nz+10)
        float *g_ex_Vy0_next,  float * g_ex_Vx0_next, float * g_ex_Vz0_next, float * g_ex_sigmayy0_next, float *g_ex_sigmaxx0_next, float * g_ex_sigmazz0_next, float * g_ex_sigmaxy0_next, float * g_ex_sigmaxz0_next, float * g_ex_sigmayz0_next,//(ny+10,nx+10,nz+10)
        float *g_ex_Vy0_pre,  float * g_ex_Vx0_pre, float * g_ex_Vz0_pre, float * g_ex_sigmayy0_pre, float *g_ex_sigmaxx0_pre, float * g_ex_sigmazz0_pre, float * g_ex_sigmaxy0_pre, float * g_ex_sigmaxz0_pre, float * g_ex_sigmayz0_pre,//(ny+10,nx+10,nz+10) 
     	float * g_ex_m1_y,    float * g_ex_m1_x,    float * g_ex_m1_z,  float * g_ex_m2, float * g_ex_m3,  float * g_ex_m2m3)//(ny+10,nx+10,nz+10) 
{

	float c1=35.0/294912.0,c2=-405.0/229376.0,c3=567.0/40960.0,c4=-735.0/8192.0,c5=19845.0/16384.0;

	//GPU thread index
	int iz, ix, iy;
	iz = blockIdx.x*blockDim.x + threadIdx.x;
	ix = blockIdx.y*blockDim.y + threadIdx.y;
	iy = blockIdx.z*blockDim.z + threadIdx.z;
	//gt = it;
        	g_ex_Vx0_pre[n3d_index_ex(iz,ix  ,iy)] = /*g_ex_Vx0_pre[n3d_index_ex(iz,ix  ,iy)]*/	+ g_ex_Vx0_next[n3d_index_ex(iz, ix, iy)]	

									+ g_ex_m2m3[n3d_index_ex(iz,ix-5, iy)]*c1*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix-5,iy)]							
							 		+ g_ex_m2m3[n3d_index_ex(iz,ix-4, iy)]*c2*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m2m3[n3d_index_ex(iz,ix-3, iy)]*c3*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m2m3[n3d_index_ex(iz,ix-2, iy)]*c4*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m2m3[n3d_index_ex(iz,ix-1, iy)]*c5*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix-1,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix+1, iy)]*c4*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix+2, iy)]*c3*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix+3, iy)]*c2*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix+4, iy)]*c1*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix+4,iy)]
	

									+ g_ex_m2[n3d_index_ex(iz,ix-5, iy)]*c1*g_ex_sigmayy0_now[n3d_index_ex(iz,ix-5,iy)]							
							 		+ g_ex_m2[n3d_index_ex(iz,ix-4, iy)]*c2*g_ex_sigmayy0_now[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m2[n3d_index_ex(iz,ix-3, iy)]*c3*g_ex_sigmayy0_now[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m2[n3d_index_ex(iz,ix-2, iy)]*c4*g_ex_sigmayy0_now[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m2[n3d_index_ex(iz,ix-1, iy)]*c5*g_ex_sigmayy0_now[n3d_index_ex(iz,ix-1,iy)]	
									- g_ex_m2[n3d_index_ex(iz,  ix, iy)]*c5*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+1, iy)]*c4*g_ex_sigmayy0_now[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+2, iy)]*c3*g_ex_sigmayy0_now[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+3, iy)]*c2*g_ex_sigmayy0_now[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+4, iy)]*c1*g_ex_sigmayy0_now[n3d_index_ex(iz,ix+4,iy)]	
	

									+ g_ex_m2[n3d_index_ex(iz,ix-5, iy)]*c1*g_ex_sigmazz0_now[n3d_index_ex(iz,ix-5,iy)]							
							 		+ g_ex_m2[n3d_index_ex(iz,ix-4, iy)]*c2*g_ex_sigmazz0_now[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m2[n3d_index_ex(iz,ix-3, iy)]*c3*g_ex_sigmazz0_now[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m2[n3d_index_ex(iz,ix-2, iy)]*c4*g_ex_sigmazz0_now[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m2[n3d_index_ex(iz,ix-1, iy)]*c5*g_ex_sigmazz0_now[n3d_index_ex(iz,ix-1,iy)]	
									- g_ex_m2[n3d_index_ex(iz,  ix, iy)]*c5*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+1, iy)]*c4*g_ex_sigmazz0_now[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+2, iy)]*c3*g_ex_sigmazz0_now[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+3, iy)]*c2*g_ex_sigmazz0_now[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+4, iy)]*c1*g_ex_sigmazz0_now[n3d_index_ex(iz,ix+4,iy)]	
	

									+ g_ex_m3[n3d_index_ex(iz,ix, iy-4)]*c1*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m3[n3d_index_ex(iz,ix, iy-3)]*c2*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy-2)]*c3*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy-1)]*c4*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix,iy-1)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+1)]*c5*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+2)]*c4*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+3)]*c3*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+4)]*c2*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix,iy+4)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+5)]*c1*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix,iy+5)]							
	

									+ g_ex_m3[n3d_index_ex(iz-4,ix, iy)]*c1*g_ex_sigmaxz0_now[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m3[n3d_index_ex(iz-3,ix, iy)]*c2*g_ex_sigmaxz0_now[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m3[n3d_index_ex(iz-2,ix, iy)]*c3*g_ex_sigmaxz0_now[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m3[n3d_index_ex(iz-1,ix, iy)]*c4*g_ex_sigmaxz0_now[n3d_index_ex(iz-1,ix,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,  ix, iy)]*c5*g_ex_sigmaxz0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+1,ix, iy)]*c5*g_ex_sigmaxz0_now[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+2,ix, iy)]*c4*g_ex_sigmaxz0_now[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+3,ix, iy)]*c3*g_ex_sigmaxz0_now[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+4,ix, iy)]*c2*g_ex_sigmaxz0_now[n3d_index_ex(iz+4,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+5,ix, iy)]*c1*g_ex_sigmaxz0_now[n3d_index_ex(iz+5,ix,iy)]	;						
	


         	g_ex_Vy0_pre[n3d_index_ex(iz,ix  ,iy)] = /*g_ex_Vy0_pre[n3d_index_ex(iz,ix  ,iy)]*/	+ g_ex_Vy0_next[n3d_index_ex(iz, ix, iy)]	

									+ g_ex_m2m3[n3d_index_ex(iz,ix, iy-5)]*c1*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy-5)]							
							 		+ g_ex_m2m3[n3d_index_ex(iz,ix, iy-4)]*c2*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m2m3[n3d_index_ex(iz,ix, iy-3)]*c3*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m2m3[n3d_index_ex(iz,ix, iy-2)]*c4*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m2m3[n3d_index_ex(iz,ix, iy-1)]*c5*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy-1)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix, iy+1)]*c4*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix, iy+2)]*c3*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix, iy+3)]*c2*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix, iy+4)]*c1*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy+4)]
	

									+ g_ex_m2[n3d_index_ex(iz,ix, iy-5)]*c1*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy-5)]							
							 		+ g_ex_m2[n3d_index_ex(iz,ix, iy-4)]*c2*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m2[n3d_index_ex(iz,ix, iy-3)]*c3*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m2[n3d_index_ex(iz,ix, iy-2)]*c4*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m2[n3d_index_ex(iz,ix, iy-1)]*c5*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy-1)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+1)]*c4*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+2)]*c3*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+3)]*c2*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+4)]*c1*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy+4)]	
	

									+ g_ex_m2[n3d_index_ex(iz,ix, iy-5)]*c1*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy-5)]							
							 		+ g_ex_m2[n3d_index_ex(iz,ix, iy-4)]*c2*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m2[n3d_index_ex(iz,ix, iy-3)]*c3*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m2[n3d_index_ex(iz,ix, iy-2)]*c4*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m2[n3d_index_ex(iz,ix, iy-1)]*c5*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy-1)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+1)]*c4*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+2)]*c3*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+3)]*c2*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+4)]*c1*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy+4)]	
	

									+ g_ex_m3[n3d_index_ex(iz-4,ix, iy)]*c1*g_ex_sigmayz0_now[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m3[n3d_index_ex(iz-3,ix, iy)]*c2*g_ex_sigmayz0_now[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m3[n3d_index_ex(iz-2,ix, iy)]*c3*g_ex_sigmayz0_now[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m3[n3d_index_ex(iz-1,ix, iy)]*c4*g_ex_sigmayz0_now[n3d_index_ex(iz-1,ix,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmayz0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+1,ix, iy)]*c5*g_ex_sigmayz0_now[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+2,ix, iy)]*c4*g_ex_sigmayz0_now[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+3,ix, iy)]*c3*g_ex_sigmayz0_now[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+4,ix, iy)]*c2*g_ex_sigmayz0_now[n3d_index_ex(iz+4,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+5,ix, iy)]*c1*g_ex_sigmayz0_now[n3d_index_ex(iz+5,ix,iy)]							
	

									+ g_ex_m3[n3d_index_ex(iz,ix-4, iy)]*c1*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m3[n3d_index_ex(iz,ix-3, iy)]*c2*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,ix-2, iy)]*c3*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,ix-1, iy)]*c4*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix-1,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+1, iy)]*c5*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+2, iy)]*c4*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+3, iy)]*c3*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+4, iy)]*c2*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix+4,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+5, iy)]*c1*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix+5,iy)]	;						




         	g_ex_Vz0_pre[n3d_index_ex(iz,ix  ,iy)] = /*g_ex_Vz0_pre[n3d_index_ex(iz,ix  ,iy)]*/	+ g_ex_Vz0_next[n3d_index_ex(iz, ix, iy)]	

									+ g_ex_m2m3[n3d_index_ex(iz-5,ix, iy)]*c1*g_ex_sigmazz0_now[n3d_index_ex(iz-5,ix,iy)]							
							 		+ g_ex_m2m3[n3d_index_ex(iz-4,ix, iy)]*c2*g_ex_sigmazz0_now[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m2m3[n3d_index_ex(iz-3,ix, iy)]*c3*g_ex_sigmazz0_now[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m2m3[n3d_index_ex(iz-2,ix, iy)]*c4*g_ex_sigmazz0_now[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m2m3[n3d_index_ex(iz-1,ix, iy)]*c5*g_ex_sigmazz0_now[n3d_index_ex(iz-1,ix,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz+1,ix, iy)]*c4*g_ex_sigmazz0_now[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz+2,ix, iy)]*c3*g_ex_sigmazz0_now[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz+3,ix, iy)]*c2*g_ex_sigmazz0_now[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz+4,ix, iy)]*c1*g_ex_sigmazz0_now[n3d_index_ex(iz+4,ix,iy)]
	

									+ g_ex_m2[n3d_index_ex(iz-5,ix, iy)]*c1*g_ex_sigmaxx0_now[n3d_index_ex(iz-5,ix,iy)]							
							 		+ g_ex_m2[n3d_index_ex(iz-4,ix, iy)]*c2*g_ex_sigmaxx0_now[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m2[n3d_index_ex(iz-3,ix, iy)]*c3*g_ex_sigmaxx0_now[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m2[n3d_index_ex(iz-2,ix, iy)]*c4*g_ex_sigmaxx0_now[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m2[n3d_index_ex(iz-1,ix, iy)]*c5*g_ex_sigmaxx0_now[n3d_index_ex(iz-1,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+1,ix, iy)]*c4*g_ex_sigmaxx0_now[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+2,ix, iy)]*c3*g_ex_sigmaxx0_now[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+3,ix, iy)]*c2*g_ex_sigmaxx0_now[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+4,ix, iy)]*c1*g_ex_sigmaxx0_now[n3d_index_ex(iz+4,ix,iy)]
	

									+ g_ex_m2[n3d_index_ex(iz-5,ix, iy)]*c1*g_ex_sigmayy0_now[n3d_index_ex(iz-5,ix,iy)]							
							 		+ g_ex_m2[n3d_index_ex(iz-4,ix, iy)]*c2*g_ex_sigmayy0_now[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m2[n3d_index_ex(iz-3,ix, iy)]*c3*g_ex_sigmayy0_now[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m2[n3d_index_ex(iz-2,ix, iy)]*c4*g_ex_sigmayy0_now[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m2[n3d_index_ex(iz-1,ix, iy)]*c5*g_ex_sigmayy0_now[n3d_index_ex(iz-1,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+1,ix, iy)]*c4*g_ex_sigmayy0_now[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+2,ix, iy)]*c3*g_ex_sigmayy0_now[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+3,ix, iy)]*c2*g_ex_sigmayy0_now[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+4,ix, iy)]*c1*g_ex_sigmayy0_now[n3d_index_ex(iz+4,ix,iy)]
	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy-4)]*c1*g_ex_sigmayz0_now[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m3[n3d_index_ex(iz,ix, iy-3)]*c2*g_ex_sigmayz0_now[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy-2)]*c3*g_ex_sigmayz0_now[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy-1)]*c4*g_ex_sigmayz0_now[n3d_index_ex(iz,ix,iy-1)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmayz0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+1)]*c5*g_ex_sigmayz0_now[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+2)]*c4*g_ex_sigmayz0_now[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+3)]*c3*g_ex_sigmayz0_now[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+4)]*c2*g_ex_sigmayz0_now[n3d_index_ex(iz,ix,iy+4)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+5)]*c1*g_ex_sigmayz0_now[n3d_index_ex(iz,ix,iy+5)]							
	

									+ g_ex_m3[n3d_index_ex(iz,ix-4, iy)]*c1*g_ex_sigmaxz0_now[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m3[n3d_index_ex(iz,ix-3, iy)]*c2*g_ex_sigmaxz0_now[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,ix-2, iy)]*c3*g_ex_sigmaxz0_now[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,ix-1, iy)]*c4*g_ex_sigmaxz0_now[n3d_index_ex(iz,ix-1,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmaxz0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+1, iy)]*c5*g_ex_sigmaxz0_now[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+2, iy)]*c4*g_ex_sigmaxz0_now[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+3, iy)]*c3*g_ex_sigmaxz0_now[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+4, iy)]*c2*g_ex_sigmaxz0_now[n3d_index_ex(iz,ix+4,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+5, iy)]*c1*g_ex_sigmaxz0_now[n3d_index_ex(iz,ix+5,iy)]	;						


		

              g_ex_sigmaxx0_pre[n3d_index_ex(iz,ix  ,iy)] =/* g_ex_sigmaxx0_pre[n3d_index_ex(iz,ix  , iy)]*/	+ g_ex_sigmaxx0_next[n3d_index_ex(iz,ix  , iy)] 
									+ g_ex_m1_x[n3d_index_ex(iz,ix-4, iy)]*c1*g_ex_Vx0_now[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m1_x[n3d_index_ex(iz,ix-3, iy)]*c2*g_ex_Vx0_now[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m1_x[n3d_index_ex(iz,ix-2, iy)]*c3*g_ex_Vx0_now[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m1_x[n3d_index_ex(iz,ix-1, iy)]*c4*g_ex_Vx0_now[n3d_index_ex(iz,ix-1,iy)]	
									+ g_ex_m1_x[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix+1, iy)]*c5*g_ex_Vx0_now[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix+2, iy)]*c4*g_ex_Vx0_now[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix+3, iy)]*c3*g_ex_Vx0_now[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix+4, iy)]*c2*g_ex_Vx0_now[n3d_index_ex(iz,ix+4,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix+5, iy)]*c1*g_ex_Vx0_now[n3d_index_ex(iz,ix+5,iy)]	;						

	    
              g_ex_sigmayy0_pre[n3d_index_ex(iz,ix  ,iy)] = /*g_ex_sigmayy0_pre[n3d_index_ex(iz,ix  , iy)]*/	+ g_ex_sigmayy0_next[n3d_index_ex(iz,ix  , iy)] 
									+ g_ex_m1_y[n3d_index_ex(iz,ix, iy-4)]*c1*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m1_y[n3d_index_ex(iz,ix, iy-3)]*c2*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m1_y[n3d_index_ex(iz,ix, iy-2)]*c3*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m1_y[n3d_index_ex(iz,ix, iy-1)]*c4*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy-1)]	
									+ g_ex_m1_y[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix, iy+1)]*c5*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix, iy+2)]*c4*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix, iy+3)]*c3*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix, iy+4)]*c2*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy+4)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix, iy+5)]*c1*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy+5)]	;		


              g_ex_sigmazz0_pre[n3d_index_ex(iz,ix  ,iy)] = /*g_ex_sigmazz0_pre[n3d_index_ex(iz,ix  , iy)]*/	+ g_ex_sigmazz0_next[n3d_index_ex(iz,ix  , iy)] 
									+ g_ex_m1_z[n3d_index_ex(iz-4,ix, iy)]*c1*g_ex_Vz0_now[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m1_z[n3d_index_ex(iz-3,ix, iy)]*c2*g_ex_Vz0_now[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m1_z[n3d_index_ex(iz-2,ix, iy)]*c3*g_ex_Vz0_now[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m1_z[n3d_index_ex(iz-1,ix, iy)]*c4*g_ex_Vz0_now[n3d_index_ex(iz-1,ix,iy)]	
									+ g_ex_m1_z[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz+1,ix, iy)]*c5*g_ex_Vz0_now[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz+2,ix, iy)]*c4*g_ex_Vz0_now[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz+3,ix, iy)]*c3*g_ex_Vz0_now[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz+4,ix, iy)]*c2*g_ex_Vz0_now[n3d_index_ex(iz+4,ix,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz+5,ix, iy)]*c1*g_ex_Vz0_now[n3d_index_ex(iz+5,ix,iy)]	;		 
	
	


              g_ex_sigmaxy0_pre[n3d_index_ex(iz,ix  ,iy)] = /*g_ex_sigmaxy0_pre[n3d_index_ex(iz,ix  , iy)]*/	+ g_ex_sigmaxy0_next[n3d_index_ex(iz,ix  , iy)] 
									+ g_ex_m1_y[n3d_index_ex(iz,ix-4, iy)]*c1*g_ex_Vy0_now[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m1_y[n3d_index_ex(iz,ix-3, iy)]*c2*g_ex_Vy0_now[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m1_y[n3d_index_ex(iz,ix-2, iy)]*c3*g_ex_Vy0_now[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m1_y[n3d_index_ex(iz,ix-1, iy)]*c4*g_ex_Vy0_now[n3d_index_ex(iz,ix-1,iy)]	
									+ g_ex_m1_y[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix+1, iy)]*c5*g_ex_Vy0_now[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix+2, iy)]*c4*g_ex_Vy0_now[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix+3, iy)]*c3*g_ex_Vy0_now[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix+4, iy)]*c2*g_ex_Vy0_now[n3d_index_ex(iz,ix+4,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix+5, iy)]*c1*g_ex_Vy0_now[n3d_index_ex(iz,ix+5,iy)]	

	    
									+ g_ex_m1_x[n3d_index_ex(iz,ix, iy-4)]*c1*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m1_x[n3d_index_ex(iz,ix, iy-3)]*c2*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m1_x[n3d_index_ex(iz,ix, iy-2)]*c3*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m1_x[n3d_index_ex(iz,ix, iy-1)]*c4*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy-1)]	
									+ g_ex_m1_x[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix, iy+1)]*c5*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix, iy+2)]*c4*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix, iy+3)]*c3*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix, iy+4)]*c2*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy+4)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix, iy+5)]*c1*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy+5)]	;		


              g_ex_sigmaxz0_pre[n3d_index_ex(iz,ix  ,iy)] = /*g_ex_sigmaxz0_pre[n3d_index_ex(iz,ix  , iy)]*/	+ g_ex_sigmaxz0_next[n3d_index_ex(iz,ix  , iy)] 
									+ g_ex_m1_x[n3d_index_ex(iz-4,ix, iy)]*c1*g_ex_Vx0_now[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m1_x[n3d_index_ex(iz-3,ix, iy)]*c2*g_ex_Vx0_now[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m1_x[n3d_index_ex(iz-2,ix, iy)]*c3*g_ex_Vx0_now[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m1_x[n3d_index_ex(iz-1,ix, iy)]*c4*g_ex_Vx0_now[n3d_index_ex(iz-1,ix,iy)]	
									+ g_ex_m1_x[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz+1,ix, iy)]*c5*g_ex_Vx0_now[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz+2,ix, iy)]*c4*g_ex_Vx0_now[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz+3,ix, iy)]*c3*g_ex_Vx0_now[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz+4,ix, iy)]*c2*g_ex_Vx0_now[n3d_index_ex(iz+4,ix,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz+5,ix, iy)]*c1*g_ex_Vx0_now[n3d_index_ex(iz+5,ix,iy)]	
							
									+ g_ex_m1_z[n3d_index_ex(iz,ix-4, iy)]*c1*g_ex_Vz0_now[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m1_z[n3d_index_ex(iz,ix-3, iy)]*c2*g_ex_Vz0_now[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m1_z[n3d_index_ex(iz,ix-2, iy)]*c3*g_ex_Vz0_now[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m1_z[n3d_index_ex(iz,ix-1, iy)]*c4*g_ex_Vz0_now[n3d_index_ex(iz,ix-1,iy)]	
									+ g_ex_m1_z[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix+1, iy)]*c5*g_ex_Vz0_now[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix+2, iy)]*c4*g_ex_Vz0_now[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix+3, iy)]*c3*g_ex_Vz0_now[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix+4, iy)]*c2*g_ex_Vz0_now[n3d_index_ex(iz,ix+4,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix+5, iy)]*c1*g_ex_Vz0_now[n3d_index_ex(iz,ix+5,iy)]	;						


              g_ex_sigmayz0_pre[n3d_index_ex(iz,ix  ,iy)] = /*g_ex_sigmayz0_pre[n3d_index_ex(iz,ix  , iy)]*/	+ g_ex_sigmayz0_next[n3d_index_ex(iz,ix  , iy)] 
									+ g_ex_m1_y[n3d_index_ex(iz-4,ix, iy)]*c1*g_ex_Vy0_now[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m1_y[n3d_index_ex(iz-3,ix, iy)]*c2*g_ex_Vy0_now[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m1_y[n3d_index_ex(iz-2,ix, iy)]*c3*g_ex_Vy0_now[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m1_y[n3d_index_ex(iz-1,ix, iy)]*c4*g_ex_Vy0_now[n3d_index_ex(iz-1,ix,iy)]	
									+ g_ex_m1_y[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz+1,ix, iy)]*c5*g_ex_Vy0_now[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz+2,ix, iy)]*c4*g_ex_Vy0_now[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz+3,ix, iy)]*c3*g_ex_Vy0_now[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz+4,ix, iy)]*c2*g_ex_Vy0_now[n3d_index_ex(iz+4,ix,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz+5,ix, iy)]*c1*g_ex_Vy0_now[n3d_index_ex(iz+5,ix,iy)]	
	
									+ g_ex_m1_z[n3d_index_ex(iz,ix, iy-4)]*c1*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m1_z[n3d_index_ex(iz,ix, iy-3)]*c2*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m1_z[n3d_index_ex(iz,ix, iy-2)]*c3*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m1_z[n3d_index_ex(iz,ix, iy-1)]*c4*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy-1)]	
									+ g_ex_m1_z[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix, iy+1)]*c5*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix, iy+2)]*c4*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix, iy+3)]*c3*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix, iy+4)]*c2*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy+4)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix, iy+5)]*c1*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy+5)]	;		

}


extern "C" void rtm_gpu_func(int ny, int nz, int nx, 
        float *ex_Vy0_now,  float * ex_Vx0_now, float * ex_Vz0_now, float * ex_sigmayy0_now, float *ex_sigmaxx0_now, float * ex_sigmazz0_now, float * ex_sigmaxy0_now, float * ex_sigmaxz0_now, float * ex_sigmayz0_now,//(ny+10, nx+10, nx+10)
        float *ex_Vy0_next,  float * ex_Vx0_next, float * ex_Vz0_next, float * ex_sigmayy0_next, float *ex_sigmaxx0_next, float * ex_sigmazz0_next, float * ex_sigmaxy0_next, float * ex_sigmaxz0_next, float * ex_sigmayz0_next,//(ny+10, nx+10, nx+10) 
        float *ex_Vy0_pre,  float * ex_Vx0_pre, float * ex_Vz0_pre, float * ex_sigmayy0_pre, float *ex_sigmaxx0_pre, float * ex_sigmazz0_pre, float * ex_sigmaxy0_pre, float * ex_sigmaxz0_pre, float * ex_sigmayz0_pre,//(ny+10, nx+10, nx+10) 
        float * ex_m1_y, float * ex_m1_x,float * ex_m1_z,float * ex_m2, float * ex_m3, float * ex_m2m3,//)//(nz+10,nx+10)
	float * debug, float * gpu_kernel_time)

{	
     	hipError_t err;
	hipEvent_t start1, start2, start3, stop1, stop2, stop3;
	float elapsedTime1 = 0.0f;//time for data copy in GPU
	float elapsedTime2 = 0.0f;//time for kernel computing + overlapping
	float elapsedTime3 = 0.0f;//time for data copy out GPU

	hipSetDevice(0);
	hipEventCreate(&start1);
	hipEventCreate(&start2);
	hipEventCreate(&start3);
	hipEventCreate(&stop1);
	hipEventCreate(&stop2);
	hipEventCreate(&stop3);
	
#ifdef MULTI_GPUS

	//config multiple GPU status
	//Acquire the number of GPU available
	int n_gpus = 1;
	int gpu_n;
	hipGetDeviceCount(&gpu_n);
	if(n_gpus>gpu_n){
		fprintf(stderr, "[EXIT] GPU number avalable %d less than number assigned %d\n", gpu_n, n_gpus);
		//fprintf(stderr, "Set GPU number from assigned %d to GPU number avalable %d \n", n_gpus, gpu_n);
		//n_gpus = gpu_n;
		exit(1);
	}

	//init GPU devices with P2P being enabled
	setup_cuda(n_gpus);

	// GPU inputs and outputs corresponds to CPU inputs and outputs
	float *g_ex_Vx0_now[n_gpus], *g_ex_Vx0_next[n_gpus], *g_ex_Vx0_pre[n_gpus];
	float *g_ex_Vy0_now[n_gpus], *g_ex_Vy0_next[n_gpus], *g_ex_Vy0_pre[n_gpus];
	float *g_ex_Vz0_now[n_gpus], *g_ex_Vz0_next[n_gpus], *g_ex_Vz0_pre[n_gpus];
	float *g_ex_sigmaxx0_now[n_gpus], *g_ex_sigmaxx0_next[n_gpus], *g_ex_sigmaxx0_pre[n_gpus];
	float *g_ex_sigmayy0_now[n_gpus], *g_ex_sigmayy0_next[n_gpus], *g_ex_sigmayy0_pre[n_gpus];
	float *g_ex_sigmazz0_now[n_gpus], *g_ex_sigmazz0_next[n_gpus], *g_ex_sigmazz0_pre[n_gpus];
	float *g_ex_sigmaxy0_now[n_gpus], *g_ex_sigmaxy0_next[n_gpus], *g_ex_sigmaxy0_pre[n_gpus];
	float *g_ex_sigmaxz0_now[n_gpus], *g_ex_sigmaxz0_next[n_gpus], *g_ex_sigmaxz0_pre[n_gpus];
	float *g_ex_sigmayz0_now[n_gpus], *g_ex_sigmayz0_next[n_gpus], *g_ex_sigmayz0_pre[n_gpus];

	float *g_ex_m2[n_gpus];
	float *g_ex_m3[n_gpus];
	float *g_ex_m2m3[n_gpus];
	float *g_ex_m1_x[n_gpus];
	float *g_ex_m1_z[n_gpus];
	float *g_ex_m1_y[n_gpus];

	
	//config index and offset

	int n1 = (NZ+10), n2 = (NX+10), n3 = (NY+10); // NZ -->NX--> NY: fast --> slow, NX, NY, NZ refers to the original index from Fortran code, and here needs to be extended by plus 10 for stencil doundary
	n3 = (n3 - 2*radius)/n_gpus + 2*radius;		
	int n_bytes_gpu = (n1*n2*n3)*sizeof(float);   //bytes length assigned for each GPU

	int start[n_gpus];				//CPU index for copy data to GPU	
	for(int i=0; i<n_gpus; i++){			//Define coordinates and offsets parameters	
		start[i] = i*(n3-2*radius) * (n1*n2);
	}

	//offset, bytes length, and block/grid config for internal part
	int offset_internal = radius*(n1*n2);		
	int n_bytes_gpu_internal = (n3-4*radius)*(n1*n2)*sizeof(float);
	dim3 dimGrid_internal(n1/TZ, n2/TX, (n3-4*radius)/TY);
	dim3 dimBlock(TZ, TX, TY);

	//offset, bytes length, and block/grid config for upper and bottom halo parts
	int offset_halo_up = 0;	
	int offset_halo_bt = (n3-3*radius)*(n1*n2);
	int n_bytes_gpu_halo = radius*(n1*n2)*sizeof(float);
	dim3 dimGrid_halo(n1/TZ, n2/TX, radius/TY);

	//offset and bytes length for output data back to CPU
	int offset_out = radius * n1*n2;
	int n_bytes_gpu_back = n1*n2*(n3-2*radius)*sizeof(float);

	//malloc data on each GPU
	for(int i=0; i<n_gpus; i++){
		hipSetDevice(device[i]);
		//hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		hipMalloc((void **) &g_ex_Vx0_now[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_Vx0_next[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_Vx0_pre[i], n_bytes_gpu);

		hipMalloc((void **) &g_ex_Vy0_now[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_Vy0_next[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_Vy0_pre[i], n_bytes_gpu);

		hipMalloc((void **) &g_ex_Vz0_now[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_Vz0_next[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_Vz0_pre[i], n_bytes_gpu);

		hipMalloc((void **) &g_ex_sigmaxx0_now[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmaxx0_next[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmaxx0_pre[i], n_bytes_gpu);

		hipMalloc((void **) &g_ex_sigmayy0_now[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmayy0_next[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmayy0_pre[i], n_bytes_gpu);

		hipMalloc((void **) &g_ex_sigmazz0_now[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmazz0_next[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmazz0_pre[i], n_bytes_gpu);

		hipMalloc((void **) &g_ex_sigmaxy0_now[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmaxy0_next[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmaxy0_pre[i], n_bytes_gpu);

		hipMalloc((void **) &g_ex_sigmaxz0_now[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmaxz0_next[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmaxz0_pre[i], n_bytes_gpu);

		hipMalloc((void **) &g_ex_sigmayz0_now[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmayz0_next[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmayz0_pre[i], n_bytes_gpu);

		hipMalloc((void **) &g_ex_m2[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_m3[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_m2m3[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_m1_x[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_m1_y[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_m1_z[i], n_bytes_gpu);
		
	}
	
	//copy data to each GPU
	hipSetDevice(0);
	hipEventRecord(start1, 0);
	for(int i = 0; i<n_gpus; i++){
		fprintf(stderr, "Copy to GPU [%d] ...\n", device[i]);
		hipSetDevice(device[i]);

		hipMemcpy(g_ex_Vx0_now[i],  &ex_Vx0_now[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_Vx0_next[i], &ex_Vx0_next[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_Vx0_pre[i], &ex_Vx0_pre[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);

		hipMemcpy(g_ex_Vy0_now[i],  &ex_Vy0_now[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_Vy0_next[i], &ex_Vy0_next[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_Vy0_pre[i], &ex_Vy0_pre[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);

		hipMemcpy(g_ex_Vz0_now[i],  &ex_Vz0_now[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_Vz0_next[i], &ex_Vz0_next[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_Vz0_pre[i], &ex_Vz0_pre[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);

		hipMemcpy(g_ex_sigmaxx0_now[i], &ex_sigmaxx0_now[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmaxx0_next[i], &ex_sigmaxx0_next[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmaxx0_pre[i], &ex_sigmaxx0_pre[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);

		hipMemcpy(g_ex_sigmayy0_now[i], &ex_sigmayy0_now[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmayy0_next[i], &ex_sigmayy0_next[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmayy0_pre[i], &ex_sigmayy0_pre[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);

		hipMemcpy(g_ex_sigmazz0_now[i], &ex_sigmazz0_now[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmazz0_next[i], &ex_sigmazz0_next[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmazz0_pre[i], &ex_sigmazz0_pre[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);

		hipMemcpy(g_ex_sigmaxy0_now[i], &ex_sigmaxy0_now[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmaxy0_next[i], &ex_sigmaxy0_next[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmaxy0_pre[i], &ex_sigmaxy0_pre[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);

		hipMemcpy(g_ex_sigmaxz0_now[i], &ex_sigmaxz0_now[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmaxz0_next[i], &ex_sigmaxz0_next[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmaxz0_pre[i], &ex_sigmaxz0_pre[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);

		hipMemcpy(g_ex_sigmayz0_now[i], &ex_sigmayz0_now[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmayz0_next[i], &ex_sigmayz0_next[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmayz0_pre[i], &ex_sigmayz0_pre[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);

		hipMemcpy(g_ex_m2[i], &ex_m2[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_m3[i], &ex_m3[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_m2m3[i], &ex_m2m3[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_m1_x[i], &ex_m1_x[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_m1_y[i], &ex_m1_y[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_m1_z[i], &ex_m1_z[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
	}
	hipSetDevice(0);
	hipEventRecord(stop1, 0);
	

	//Define separate streams for overlapping communication
	hipStream_t stream_halo[n_gpus], stream_internal[n_gpus];
	for(int i=0; i<n_gpus; i++){
		hipSetDevice(device[i]);
		hipStreamCreate(&stream_halo[i]);
		hipStreamCreate(&stream_internal[i]);
	}

	fprintf(stderr,"GPU Computing ... ...(NZ=%d, NX=%d, NY=%d, TZ=%d, TX=%d, TY=%d)\n", nz, nx, ny, TZ, TX, TY);
	
	hipSetDevice(0);
	hipEventRecord(start2, 0);

	for(int g_it=0; g_it<Steps_write_back;g_it++){
	
	//Calculate the halo regions first
	for(int i=0; i<n_gpus; i++){
		hipSetDevice(i);
		
		fprintf(stderr, "Upper halo computing, on GPU [%d]\n", i);
		rtm_gpu_kernel<<<dimGrid_halo, dimBlock,0,stream_halo[i]>>>(ny, nz, nx,
			g_ex_Vy0_now[i] + offset_halo_up, g_ex_Vx0_now[i] + offset_halo_up, g_ex_Vz0_now[i] + offset_halo_up, g_ex_sigmayy0_now[i] + offset_halo_up, g_ex_sigmaxx0_now[i] + offset_halo_up, g_ex_sigmazz0_now[i] + offset_halo_up, g_ex_sigmaxy0_now[i] + offset_halo_up, g_ex_sigmaxz0_now[i] + offset_halo_up, g_ex_sigmayz0_now[i] + offset_halo_up,
			g_ex_Vy0_next[i] + offset_halo_up, g_ex_Vx0_next[i] + offset_halo_up, g_ex_Vz0_next[i] + offset_halo_up, g_ex_sigmayy0_next[i] + offset_halo_up, g_ex_sigmaxx0_next[i] + offset_halo_up, g_ex_sigmazz0_next[i] + offset_halo_up, g_ex_sigmaxy0_next[i] + offset_halo_up, g_ex_sigmaxz0_next[i] + offset_halo_up, g_ex_sigmayz0_next[i] + offset_halo_up,
			g_ex_Vy0_pre[i] + offset_halo_up, g_ex_Vx0_pre[i] + offset_halo_up, g_ex_Vz0_pre[i] + offset_halo_up, g_ex_sigmayy0_pre[i] + offset_halo_up, g_ex_sigmaxx0_pre[i] + offset_halo_up, g_ex_sigmazz0_pre[i] + offset_halo_up, g_ex_sigmaxy0_pre[i] + offset_halo_up, g_ex_sigmaxz0_pre[i] + offset_halo_up, g_ex_sigmayz0_pre[i] + offset_halo_up,
			g_ex_m1_y[i] + offset_halo_up, g_ex_m1_x[i] + offset_halo_up, g_ex_m1_z[i] + offset_halo_up, g_ex_m2[i] + offset_halo_up, g_ex_m3[i] + offset_halo_up, g_ex_m2m3[i] + offset_halo_up);

		fprintf(stderr, "Bottom halo computing, on GPU [%d]\n", i);
		rtm_gpu_kernel<<<dimGrid_halo, dimBlock,0,stream_halo[i]>>>(ny, nz, nx,
			g_ex_Vy0_now[i] + offset_halo_bt, g_ex_Vx0_now[i] + offset_halo_bt, g_ex_Vz0_now[i] + offset_halo_bt, g_ex_sigmayy0_now[i] + offset_halo_bt, g_ex_sigmaxx0_now[i] + offset_halo_bt, g_ex_sigmazz0_now[i] + offset_halo_bt, g_ex_sigmaxy0_now[i] + offset_halo_bt, g_ex_sigmaxz0_now[i] + offset_halo_bt, g_ex_sigmayz0_now[i] + offset_halo_bt,
			g_ex_Vy0_next[i] + offset_halo_bt, g_ex_Vx0_next[i] + offset_halo_bt, g_ex_Vz0_next[i] + offset_halo_bt, g_ex_sigmayy0_next[i] + offset_halo_bt, g_ex_sigmaxx0_next[i] + offset_halo_bt, g_ex_sigmazz0_next[i] + offset_halo_bt, g_ex_sigmaxy0_next[i] + offset_halo_bt, g_ex_sigmaxz0_next[i] + offset_halo_bt, g_ex_sigmayz0_next[i] + offset_halo_bt,
			g_ex_Vy0_pre[i] + offset_halo_bt, g_ex_Vx0_pre[i] + offset_halo_bt, g_ex_Vz0_pre[i] + offset_halo_bt, g_ex_sigmayy0_pre[i] + offset_halo_bt, g_ex_sigmaxx0_pre[i] + offset_halo_bt, g_ex_sigmazz0_pre[i] + offset_halo_bt, g_ex_sigmaxy0_pre[i] + offset_halo_bt, g_ex_sigmaxz0_pre[i] + offset_halo_bt, g_ex_sigmayz0_pre[i] + offset_halo_bt,
			g_ex_m1_y[i] + offset_halo_bt, g_ex_m1_x[i] + offset_halo_bt, g_ex_m1_z[i] + offset_halo_bt, g_ex_m2[i] + offset_halo_bt, g_ex_m3[i] + offset_halo_bt, g_ex_m2m3[i] + offset_halo_bt);

		hipStreamQuery(stream_halo[i]);
		//}
	}	

	//Compute the internal part
	for(int i=0; i<n_gpus; i++){

		hipSetDevice(i);	
		fprintf(stderr, "Internal Computing, GPU [%d]\n", i);
		rtm_gpu_kernel<<<dimGrid_internal, dimBlock, 0, stream_internal[i]>>>(ny, nz, nx,
			g_ex_Vy0_now[i] + offset_internal, g_ex_Vx0_now[i] + offset_internal, g_ex_Vz0_now[i] + offset_internal, g_ex_sigmayy0_now[i] + offset_internal, g_ex_sigmaxx0_now[i] + offset_internal, g_ex_sigmazz0_now[i] + offset_internal, g_ex_sigmaxy0_now[i] + offset_internal, g_ex_sigmaxz0_now[i] + offset_internal, g_ex_sigmayz0_now[i] + offset_internal,
			g_ex_Vy0_next[i] + offset_internal, g_ex_Vx0_next[i] + offset_internal, g_ex_Vz0_next[i] + offset_internal, g_ex_sigmayy0_next[i] + offset_internal, g_ex_sigmaxx0_next[i] + offset_internal, g_ex_sigmazz0_next[i] + offset_internal, g_ex_sigmaxy0_next[i] + offset_internal, g_ex_sigmaxz0_next[i] + offset_internal, g_ex_sigmayz0_next[i] + offset_internal,
		g_ex_Vy0_pre[i] + offset_internal, g_ex_Vx0_pre[i] + offset_internal, g_ex_Vz0_pre[i] + offset_internal, g_ex_sigmayy0_pre[i] + offset_internal, g_ex_sigmaxx0_pre[i] + offset_internal, g_ex_sigmazz0_pre[i] + offset_internal, g_ex_sigmaxy0_pre[i] + offset_internal, g_ex_sigmaxz0_pre[i] + offset_internal, g_ex_sigmayz0_pre[i] + offset_internal,
			g_ex_m1_y[i] + offset_internal, g_ex_m1_x[i] + offset_internal, g_ex_m1_z[i] + offset_internal, g_ex_m2[i] + offset_internal, g_ex_m3[i] + offset_internal, g_ex_m2m3[i] + offset_internal);
		}

	//Halo updating
	if(g_it < Steps_write_back-1){
		
	fprintf(stderr, "Halo Updating\n");
	//offset for each halo parts
	int offset_down_snd = (n3-2*radius)*(n1*n2);
	int offset_down_rcv = 0;//n3-2*radius)*(n1*n2);
	int offset_up_snd   = radius*n1*n2;
	int offset_up_rcv   = (n3-radius)*n1*n2;
	
	//Send halos downwards
	for(int i=0; i<n_gpus-1; i++){
		hipMemcpyPeerAsync(g_ex_Vx0_pre[i+1]+offset_down_rcv,i+1, g_ex_Vx0_pre[i]+offset_down_snd, i, n_bytes_gpu_halo, stream_halo[i]);
		hipMemcpyPeerAsync(g_ex_Vy0_pre[i+1]+offset_down_rcv,i+1, g_ex_Vy0_pre[i]+offset_down_snd, i, n_bytes_gpu_halo, stream_halo[i]);
		hipMemcpyPeerAsync(g_ex_Vz0_pre[i+1]+offset_down_rcv,i+1, g_ex_Vz0_pre[i]+offset_down_snd, i, n_bytes_gpu_halo, stream_halo[i]);
		hipMemcpyPeerAsync(g_ex_sigmaxx0_pre[i+1]+offset_down_rcv,i+1, g_ex_sigmaxx0_pre[i]+offset_down_snd, i, n_bytes_gpu_halo, stream_halo[i]);
		hipMemcpyPeerAsync(g_ex_sigmayy0_pre[i+1]+offset_down_rcv,i+1, g_ex_sigmayy0_pre[i]+offset_down_snd, i, n_bytes_gpu_halo, stream_halo[i]);
		hipMemcpyPeerAsync(g_ex_sigmazz0_pre[i+1]+offset_down_rcv,i+1, g_ex_sigmazz0_pre[i]+offset_down_snd, i, n_bytes_gpu_halo, stream_halo[i]);
		hipMemcpyPeerAsync(g_ex_sigmaxy0_pre[i+1]+offset_down_rcv,i+1, g_ex_sigmaxy0_pre[i]+offset_down_snd, i, n_bytes_gpu_halo, stream_halo[i]);
		hipMemcpyPeerAsync(g_ex_sigmaxz0_pre[i+1]+offset_down_rcv,i+1, g_ex_sigmaxz0_pre[i]+offset_down_snd, i, n_bytes_gpu_halo, stream_halo[i]);
		hipMemcpyPeerAsync(g_ex_sigmayz0_pre[i+1]+offset_down_rcv,i+1, g_ex_sigmayz0_pre[i]+offset_down_snd, i, n_bytes_gpu_halo, stream_halo[i]);
	}

	//Synchronize to avoid stalling
	for(int i=0; i<n_gpus; i++){
		hipSetDevice(i);
		hipStreamSynchronize(stream_halo[i]);
	}	

	//Send halos upwards
	for(int i=1; i<n_gpus; i++){
		hipMemcpyPeerAsync(g_ex_Vx0_pre[i-1]+offset_up_rcv,i-1, g_ex_Vx0_pre[i]+offset_up_snd, i, n_bytes_gpu_halo, stream_halo[i]);
		hipMemcpyPeerAsync(g_ex_Vy0_pre[i-1]+offset_up_rcv,i-1, g_ex_Vy0_pre[i]+offset_up_snd, i, n_bytes_gpu_halo, stream_halo[i]);
		hipMemcpyPeerAsync(g_ex_Vz0_pre[i-1]+offset_up_rcv,i-1, g_ex_Vz0_pre[i]+offset_up_snd, i, n_bytes_gpu_halo, stream_halo[i]);
		hipMemcpyPeerAsync(g_ex_sigmaxx0_pre[i-1]+offset_up_rcv,i-1, g_ex_sigmaxx0_pre[i]+offset_up_snd, i, n_bytes_gpu_halo, stream_halo[i]);
		hipMemcpyPeerAsync(g_ex_sigmayy0_pre[i-1]+offset_up_rcv,i-1, g_ex_sigmayy0_pre[i]+offset_up_snd, i, n_bytes_gpu_halo, stream_halo[i]);
		hipMemcpyPeerAsync(g_ex_sigmazz0_pre[i-1]+offset_up_rcv,i-1, g_ex_sigmazz0_pre[i]+offset_up_snd, i, n_bytes_gpu_halo, stream_halo[i]);
		hipMemcpyPeerAsync(g_ex_sigmaxz0_pre[i-1]+offset_up_rcv,i-1, g_ex_sigmaxz0_pre[i]+offset_up_snd, i, n_bytes_gpu_halo, stream_halo[i]);
		hipMemcpyPeerAsync(g_ex_sigmayz0_pre[i-1]+offset_up_rcv,i-1, g_ex_sigmayz0_pre[i]+offset_up_snd, i, n_bytes_gpu_halo, stream_halo[i]);
		hipMemcpyPeerAsync(g_ex_sigmaxy0_pre[i-1]+offset_up_rcv,i-1, g_ex_sigmaxy0_pre[i]+offset_up_snd, i, n_bytes_gpu_halo, stream_halo[i]);
	}
		


		//change pointer

		fprintf(stderr, "GPU pointer changed\n");
		float *g_tmp = NULL;
		
		for(int i=0; i<n_gpus; i++){
			hipSetDevice(device[i]);

			g_tmp = g_ex_Vx0_pre[i]; g_ex_Vx0_pre[i] = g_ex_Vx0_now[i]; g_ex_Vx0_now[i] = g_tmp;
			g_tmp = g_ex_Vx0_pre[i]; g_ex_Vx0_pre[i] = g_ex_Vx0_next[i]; g_ex_Vx0_next[i] = g_tmp; 

			g_tmp = g_ex_Vy0_pre[i]; g_ex_Vy0_pre[i] = g_ex_Vy0_now[i]; g_ex_Vy0_now[i] = g_tmp;
			g_tmp = g_ex_Vy0_pre[i]; g_ex_Vy0_pre[i] = g_ex_Vy0_next[i]; g_ex_Vy0_next[i] = g_tmp; 

			g_tmp = g_ex_Vz0_pre[i]; g_ex_Vz0_pre[i] = g_ex_Vz0_now[i]; g_ex_Vz0_now[i] = g_tmp;
			g_tmp = g_ex_Vz0_pre[i]; g_ex_Vz0_pre[i] = g_ex_Vz0_next[i]; g_ex_Vz0_next[i] = g_tmp;
 
			g_tmp = g_ex_sigmaxx0_pre[i]; g_ex_sigmaxx0_pre[i] = g_ex_sigmaxx0_now[i]; g_ex_sigmaxx0_now[i] = g_tmp;
			g_tmp = g_ex_sigmaxx0_pre[i]; g_ex_sigmaxx0_pre[i] = g_ex_sigmaxx0_next[i]; g_ex_sigmaxx0_next[i] = g_tmp;
 
			g_tmp = g_ex_sigmayy0_pre[i]; g_ex_sigmayy0_pre[i] = g_ex_sigmayy0_now[i]; g_ex_sigmayy0_now[i] = g_tmp;
			g_tmp = g_ex_sigmayy0_pre[i]; g_ex_sigmayy0_pre[i] = g_ex_sigmayy0_next[i]; g_ex_sigmayy0_next[i] = g_tmp;
 
			g_tmp = g_ex_sigmazz0_pre[i]; g_ex_sigmazz0_pre[i] = g_ex_sigmazz0_now[i]; g_ex_sigmazz0_now[i] = g_tmp;
			g_tmp = g_ex_sigmazz0_pre[i]; g_ex_sigmazz0_pre[i] = g_ex_sigmazz0_next[i]; g_ex_sigmazz0_next[i] = g_tmp;
 
			g_tmp = g_ex_sigmaxy0_pre[i]; g_ex_sigmaxy0_pre[i] = g_ex_sigmaxy0_now[i]; g_ex_sigmaxy0_now[i] = g_tmp;
			g_tmp = g_ex_sigmaxy0_pre[i]; g_ex_sigmaxy0_pre[i] = g_ex_sigmaxy0_next[i]; g_ex_sigmaxy0_next[i] = g_tmp; 

			g_tmp = g_ex_sigmaxz0_pre[i]; g_ex_sigmaxz0_pre[i] = g_ex_sigmaxz0_now[i]; g_ex_sigmaxz0_now[i] = g_tmp;
			g_tmp = g_ex_sigmaxz0_pre[i]; g_ex_sigmaxz0_pre[i] = g_ex_sigmaxz0_next[i]; g_ex_sigmaxz0_next[i] = g_tmp;
 
			g_tmp = g_ex_sigmayz0_pre[i]; g_ex_sigmayz0_pre[i] = g_ex_sigmayz0_now[i]; g_ex_sigmayz0_now[i] = g_tmp;
			g_tmp = g_ex_sigmayz0_pre[i]; g_ex_sigmayz0_pre[i] = g_ex_sigmayz0_next[i]; g_ex_sigmayz0_next[i] = g_tmp; 

		}
		}
	}
	
	hipSetDevice(0);
	hipEventRecord(stop2, 0);


	hipEventRecord(start3, 0);
	//Copy out data back to CPU
	for(int i = 0; i<n_gpus; i++){
		fprintf(stderr, "Copy to CPU [%d} ...\n", device[i]);
		hipSetDevice(device[i]);
		hipMemcpy(&ex_Vx0_pre[start[i]]+offset_out, g_ex_Vx0_pre[i]+offset_out, n_bytes_gpu_back, hipMemcpyDeviceToHost);
		hipMemcpy(&ex_Vy0_pre[start[i]]+offset_out, g_ex_Vy0_pre[i]+offset_out, n_bytes_gpu_back, hipMemcpyDeviceToHost);
		hipMemcpy(&ex_Vz0_pre[start[i]]+offset_out, g_ex_Vz0_pre[i]+offset_out, n_bytes_gpu_back, hipMemcpyDeviceToHost);
		hipMemcpy(&ex_sigmaxx0_pre[start[i]]+offset_out, g_ex_sigmaxx0_pre[i]+offset_out, n_bytes_gpu_back, hipMemcpyDeviceToHost);
		hipMemcpy(&ex_sigmayy0_pre[start[i]]+offset_out, g_ex_sigmayy0_pre[i]+offset_out, n_bytes_gpu_back, hipMemcpyDeviceToHost);
		hipMemcpy(&ex_sigmazz0_pre[start[i]]+offset_out, g_ex_sigmazz0_pre[i]+offset_out, n_bytes_gpu_back, hipMemcpyDeviceToHost);
		hipMemcpy(&ex_sigmaxy0_pre[start[i]]+offset_out, g_ex_sigmaxy0_pre[i]+offset_out, n_bytes_gpu_back, hipMemcpyDeviceToHost);
		hipMemcpy(&ex_sigmayz0_pre[start[i]]+offset_out, g_ex_sigmayz0_pre[i]+offset_out, n_bytes_gpu_back, hipMemcpyDeviceToHost);
		hipMemcpy(&ex_sigmaxz0_pre[start[i]]+offset_out, g_ex_sigmaxz0_pre[i]+offset_out, n_bytes_gpu_back, hipMemcpyDeviceToHost);
	}
	hipSetDevice(0);
	hipEventRecord(stop3, 0);

	hipEventSynchronize(stop1);
	hipEventSynchronize(stop2);
	hipEventSynchronize(stop3);

	hipEventElapsedTime(&elapsedTime1, start1, stop1);
	hipEventElapsedTime(&elapsedTime2, start2, stop2);
	hipEventElapsedTime(&elapsedTime3, start3, stop3);

	gpu_kernel_time[0] = (float)(elapsedTime1/1000.);
	gpu_kernel_time[1] = (float)(elapsedTime2/1000.);
	gpu_kernel_time[2] = (float)(elapsedTime3/1000.);
	
	hipEventDestroy(start1);
	hipEventDestroy(start2);
	hipEventDestroy(start3);
	hipEventDestroy(stop1);
	hipEventDestroy(stop2);
	hipEventDestroy(stop3);
	
	for(int i=0; i<n_gpus; i++){
		hipSetDevice(i);

		hipFree(g_ex_Vx0_now[i]);
		hipFree(g_ex_Vz0_now[i]);
		hipFree(g_ex_Vy0_now[i]);
		hipFree(g_ex_sigmaxx0_now[i]);
		hipFree(g_ex_sigmazz0_now[i]);
		hipFree(g_ex_sigmayy0_now[i]);
		hipFree(g_ex_sigmaxy0_now[i]);
		hipFree(g_ex_sigmaxz0_now[i]);
		hipFree(g_ex_sigmayz0_now[i]);
		
		//Time step +2
		hipFree(g_ex_Vx0_next[i]);
		hipFree(g_ex_Vz0_next[i]);
		hipFree(g_ex_Vy0_next[i]);
		hipFree(g_ex_sigmaxx0_next[i]);
		hipFree(g_ex_sigmazz0_next[i]);
		hipFree(g_ex_sigmayy0_next[i]);
		hipFree(g_ex_sigmaxy0_next[i]);
		hipFree(g_ex_sigmaxz0_next[i]);
		hipFree(g_ex_sigmayz0_next[i]);
	
	
		//time step 0 and output
		hipFree(g_ex_Vx0_pre[i]);
		hipFree(g_ex_Vz0_pre[i]);
		hipFree(g_ex_Vy0_pre[i]);
		hipFree(g_ex_sigmaxx0_pre[i]);
		hipFree(g_ex_sigmazz0_pre[i]);
		hipFree(g_ex_sigmayy0_pre[i]);
		hipFree(g_ex_sigmaxy0_pre[i]);
		hipFree(g_ex_sigmaxz0_pre[i]);
		hipFree(g_ex_sigmayz0_pre[i]);
	   
		//expaned arrays to store different Operators 
		hipFree(g_ex_m2[i]);
		hipFree(g_ex_m3[i]);
		hipFree(g_ex_m2m3[i]);
		hipFree(g_ex_m1_x[i]);
		hipFree(g_ex_m1_y[i]);
		hipFree(g_ex_m1_z[i]);
		}
	
#else
	//-----------blew are old code------------


	rtm_gpu_init(ny,nz,nx, 4);
		
	//time record

	//data copy in 
     	rtm_gpu_copy_in(ny, nz, nx, 
			ex_Vy0_now, ex_Vx0_now, ex_Vz0_now, ex_sigmayy0_now, ex_sigmaxx0_now, ex_sigmazz0_now, ex_sigmaxy0_now, ex_sigmaxz0_now, ex_sigmayz0_now,
			ex_Vy0_next, ex_Vx0_next, ex_Vz0_next, ex_sigmayy0_next, ex_sigmaxx0_next, ex_sigmazz0_next, ex_sigmaxy0_next, ex_sigmaxz0_next, ex_sigmayz0_next,
			ex_Vy0_pre, ex_Vx0_pre, ex_Vz0_pre, ex_sigmayy0_pre, ex_sigmaxx0_pre, ex_sigmazz0_pre, ex_sigmaxy0_pre, ex_sigmaxz0_pre, ex_sigmayz0_pre,
			ex_m1_y, ex_m1_x, ex_m1_z, ex_m2, ex_m3, ex_m2m3);
	hipEventRecord(stop1, 0);
	
	
	err = hipGetLastError();
	if(hipSuccess != err){
		fprintf(stderr, "Cuda error4: %s.\n", hipGetErrorString(err));
		exit(0);
	}	
	
	//RTM computing


	dim3 dimGrid(nz/TZ, nx/TX, ny/TY);
	dim3 dimBlock(TZ, TX, TY);


	hipEventRecord(start2, 0);
	
	fprintf(stderr,"GPU Computing ... ...(NZ=%d, NX=%d, NY=%d, TZ=%d, TX=%d, TY=%d)\n", nz, nx, ny, TZ, TX, TY);
	
	for(g_it = 0; g_it < Steps_write_back; g_it++){
		
		fprintf(stderr, "Step %d\n", g_it);
		rtm_gpu_kernel<<<dimGrid, dimBlock>>>(ny, nz, nx,
			g_ex_Vy0_now, g_ex_Vx0_now, g_ex_Vz0_now, g_ex_sigmayy0_now, g_ex_sigmaxx0_now, g_ex_sigmazz0_now, g_ex_sigmaxy0_now, g_ex_sigmaxz0_now, g_ex_sigmayz0_now,
			g_ex_Vy0_next, g_ex_Vx0_next, g_ex_Vz0_next, g_ex_sigmayy0_next, g_ex_sigmaxx0_next, g_ex_sigmazz0_next, g_ex_sigmaxy0_next, g_ex_sigmaxz0_next, g_ex_sigmayz0_next,
			g_ex_Vy0_pre, g_ex_Vx0_pre, g_ex_Vz0_pre, g_ex_sigmayy0_pre, g_ex_sigmaxx0_pre, g_ex_sigmazz0_pre, g_ex_sigmaxy0_pre, g_ex_sigmaxz0_pre, g_ex_sigmayz0_pre,
			g_ex_m1_y, g_ex_m1_x, g_ex_m1_z, g_ex_m2, g_ex_m3, g_ex_m2m3);
			//hipDeviceSynchronize();

		err = hipGetLastError();
		if(hipSuccess != err){
			fprintf(stderr, "Cuda error5: %s.\n", hipGetErrorString(err));
			exit(0);
			}
	
		if(g_it<Steps_write_back-1)	rtm_gpu_change_pointer();	
	}
	hipEventRecord(stop2, 0);
	

	//data copy out
	hipEventRecord(start3, 0);
	
	rtm_gpu_copy_out(ny, nz, nx,	
			ex_Vy0_pre, ex_Vx0_pre, ex_Vz0_pre, ex_sigmayy0_pre, ex_sigmaxx0_pre, ex_sigmazz0_pre, ex_sigmaxy0_pre, ex_sigmaxz0_pre, ex_sigmayz0_pre);
	hipEventRecord(stop3, 0);

	err = hipGetLastError();
	if(hipSuccess != err){
		fprintf(stderr, "Cuda error6: %s.\n", hipGetErrorString(err));
	}	


	//hipEventRecord(stop, 0);

	hipEventSynchronize(stop1);
	hipEventSynchronize(stop2);
	hipEventSynchronize(stop3);
	hipEventElapsedTime(&elapsedTime1, start1, stop1);
	hipEventElapsedTime(&elapsedTime2, start2, stop2);
	hipEventElapsedTime(&elapsedTime3, start3, stop3);

	gpu_kernel_time[0] = (float)(elapsedTime1/1000.);
	gpu_kernel_time[1] = (float)(elapsedTime2/1000.);
	gpu_kernel_time[2] = (float)(elapsedTime3/1000.);
	
	rtm_gpu_final();

	
	fprintf(stderr, "GPU copy in Time: %.4f\n", (float)elapsedTime1/1000.);
	fprintf(stderr, "GPU Comput. Time: %.4f\n", (float)elapsedTime2/1000.);
	fprintf(stderr, "GPU copy ot Time: %.4f\n", (float)elapsedTime3/1000.);
#endif
}


__global__ void rtm_gpu_kernel_all_shared(int it,int nt, int nz, int nx,
        float * g_ex_Vx0, float * g_ex_Vz0, float * g_ex_sigmaxx0, float * g_ex_sigmazz0, float * g_ex_sigmaxz0, //(nz, nx, nt)
        float * g_ex_m1_x,float * g_ex_m1_z,float * g_ex_aux_m2_c, float * g_ex_aux_m3_c, float * g_ex_aux_m2m3_c)//(nz+10,	nx+10)
{

	float c1=35.0/294912.0,c2=-405.0/229376.0,c3=567.0/40960.0,c4=-735.0/8192.0,c5=19845.0/16384.0;

	//GPU thread index
	int iz, ix;
	iz = blockIdx.x*blockDim.x + threadIdx.x;
	ix = blockIdx.y*blockDim.y + threadIdx.y;
	//gt = it;
 	
	__shared__ float sh_ex_aux_m2m3_c[(TZ+10)*(TX+10)];
	__shared__ float sh_ex_aux_m2_c[(TZ+10)*(TX+10)];
	__shared__ float sh_ex_aux_m3_c[(TZ+10)*(TX+10)];
	__shared__ float sh_ex_m1_x[(TZ+10)*(TX+10)];
	__shared__ float sh_ex_m1_z[(TZ+10)*(TX+10)];


	__shared__ float sh_ex_Vx0[(TZ+10)*(TX+10)];
	__shared__ float sh_ex_Vz0[(TZ+10)*(TX+10)];
	__shared__ float sh_ex_sigmaxx0[(TZ+10)*(TX+10)];
	__shared__ float sh_ex_sigmazz0[(TZ+10)*(TX+10)];
	__shared__ float sh_ex_sigmaxz0[(TZ+10)*(TX+10)];

	//sh_ex_aux_m2m3_c[threadIdx][];

	sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y)] = g_ex_aux_m2m3_c[index_ex(iz,ix)];
	sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y)] = g_ex_aux_m2_c[index_ex(iz,ix)];
	sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y)] = g_ex_aux_m3_c[index_ex(iz,ix)];
	sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y)] = g_ex_m1_x[index_ex(iz,ix)];
	sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y)] = g_ex_m1_z[index_ex(iz,ix)];

	sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y)] = g_ex_Vx0[index3d_ex(iz,ix,it+1)];
	sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y)] = g_ex_Vz0[index3d_ex(iz,ix,it+1)];
	sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y)] = g_ex_sigmaxx0[index3d_ex(iz,ix,it+1)];
	sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y)] = g_ex_sigmazz0[index3d_ex(iz,ix,it+1)];
	sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y)] = g_ex_sigmaxz0[index3d_ex(iz,ix,it+1)];


	if(threadIdx.x<5){
	sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x-5,threadIdx.y)] = g_ex_aux_m2m3_c[index_ex(iz-5,ix)];
	sh_ex_aux_m2_c[index_blk_ex(threadIdx.x-5,threadIdx.y)] = g_ex_aux_m2_c[index_ex(iz-5,ix)];
	sh_ex_aux_m3_c[index_blk_ex(threadIdx.x-5,threadIdx.y)] = g_ex_aux_m3_c[index_ex(iz-5,ix)];
	sh_ex_m1_x[index_blk_ex(threadIdx.x-5,threadIdx.y)] = g_ex_m1_x[index_ex(iz-5,ix)];
	sh_ex_m1_z[index_blk_ex(threadIdx.x-5,threadIdx.y)] = g_ex_m1_z[index_ex(iz-5,ix)];

	sh_ex_Vx0[index_blk_ex(threadIdx.x-5,threadIdx.y)] = g_ex_Vx0[index3d_ex(iz-5,ix,it+1)];
	sh_ex_Vz0[index_blk_ex(threadIdx.x-5,threadIdx.y)] = g_ex_Vz0[index3d_ex(iz-5,ix,it+1)];
	sh_ex_sigmaxx0[index_blk_ex(threadIdx.x-5,threadIdx.y)] = g_ex_sigmaxx0[index3d_ex(iz-5,ix,it+1)];
	sh_ex_sigmazz0[index_blk_ex(threadIdx.x-5,threadIdx.y)] = g_ex_sigmazz0[index3d_ex(iz-5,ix,it+1)];
	sh_ex_sigmaxz0[index_blk_ex(threadIdx.x-5,threadIdx.y)] = g_ex_sigmaxz0[index3d_ex(iz-5,ix,it+1)];
	}

	if(threadIdx.x>=TZ-5){
	sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x+5,threadIdx.y)] = g_ex_aux_m2m3_c[index_ex(iz+5,ix)];
	sh_ex_aux_m2_c[index_blk_ex(threadIdx.x+5,threadIdx.y)] = g_ex_aux_m2_c[index_ex(iz+5,ix)];
	sh_ex_aux_m3_c[index_blk_ex(threadIdx.x+5,threadIdx.y)] = g_ex_aux_m3_c[index_ex(iz+5,ix)];
	sh_ex_m1_x[index_blk_ex(threadIdx.x+5,threadIdx.y)] = g_ex_m1_x[index_ex(iz+5,ix)];
	sh_ex_m1_z[index_blk_ex(threadIdx.x+5,threadIdx.y)] = g_ex_m1_z[index_ex(iz+5,ix)];
	
	sh_ex_Vx0[index_blk_ex(threadIdx.x+5,threadIdx.y)] = g_ex_Vx0[index3d_ex(iz+5,ix,it+1)];
	sh_ex_Vz0[index_blk_ex(threadIdx.x+5,threadIdx.y)] = g_ex_Vz0[index3d_ex(iz+5,ix,it+1)];
	sh_ex_sigmaxx0[index_blk_ex(threadIdx.x+5,threadIdx.y)] = g_ex_sigmaxx0[index3d_ex(iz+5,ix,it+1)];
	sh_ex_sigmazz0[index_blk_ex(threadIdx.x+5,threadIdx.y)] = g_ex_sigmazz0[index3d_ex(iz+5,ix,it+1)];
	sh_ex_sigmaxz0[index_blk_ex(threadIdx.x+5,threadIdx.y)] = g_ex_sigmaxz0[index3d_ex(iz+5,ix,it+1)];
	}
	

	if(threadIdx.y<5){
	sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y-5)] = g_ex_aux_m2m3_c[index_ex(iz,ix-5)];
	sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y-5)] = g_ex_aux_m2_c[index_ex(iz,ix-5)];
	sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y-5)] = g_ex_aux_m3_c[index_ex(iz,ix-5)];
	sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y-5)] = g_ex_m1_x[index_ex(iz,ix-5)];
	sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y-5)] = g_ex_m1_z[index_ex(iz,ix-5)];

	sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y-5)] = g_ex_Vx0[index3d_ex(iz,ix-5,it+1)];
	sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y-5)] = g_ex_Vz0[index3d_ex(iz,ix-5,it+1)];
	sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y-5)] = g_ex_sigmaxx0[index3d_ex(iz,ix-5,it+1)];
	sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y-5)] = g_ex_sigmazz0[index3d_ex(iz,ix-5,it+1)];
	sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y-5)] = g_ex_sigmaxz0[index3d_ex(iz,ix-5,it+1)];
	}


	if(threadIdx.y>=TX-5){
	sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y+5)] = g_ex_aux_m2m3_c[index_ex(iz,ix+5)];
	sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y+5)] = g_ex_aux_m2_c[index_ex(iz,ix+5)];
	sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y+5)] = g_ex_aux_m3_c[index_ex(iz,ix+5)];
	sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y+5)] = g_ex_m1_x[index_ex(iz,ix+5)];
	sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y+5)] = g_ex_m1_z[index_ex(iz,ix+5)];

	sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y+5)] = g_ex_Vx0[index3d_ex(iz,ix+5,it+1)];
	sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y+5)] = g_ex_Vz0[index3d_ex(iz,ix+5,it+1)];
	sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y+5)] = g_ex_sigmaxx0[index3d_ex(iz,ix+5,it+1)];
	sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y+5)] = g_ex_sigmazz0[index3d_ex(iz,ix+5,it+1)];
	sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y+5)] = g_ex_sigmaxz0[index3d_ex(iz,ix+5,it+1)];
	}



	if(threadIdx.x <5 && threadIdx.y <5){
	sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x-5,threadIdx.y-5)] = g_ex_aux_m2m3_c[index_ex(iz-5,ix-5)];
	sh_ex_aux_m2_c[index_blk_ex(threadIdx.x-5,threadIdx.y-5)] = g_ex_aux_m2_c[index_ex(iz-5,ix-5)];
	sh_ex_aux_m3_c[index_blk_ex(threadIdx.x-5,threadIdx.y-5)] = g_ex_aux_m3_c[index_ex(iz-5,ix-5)];
	sh_ex_m1_x[index_blk_ex(threadIdx.x-5,threadIdx.y-5)] = g_ex_m1_x[index_ex(iz-5,ix-5)];
	sh_ex_m1_z[index_blk_ex(threadIdx.x-5,threadIdx.y-5)] = g_ex_m1_z[index_ex(iz-5,ix-5)];

	sh_ex_Vx0[index_blk_ex(threadIdx.x-5,threadIdx.y-5)] = g_ex_Vx0[index3d_ex(iz-5,ix-5,it+1)];
	sh_ex_Vz0[index_blk_ex(threadIdx.x-5,threadIdx.y-5)] = g_ex_Vz0[index3d_ex(iz-5,ix-5,it+1)];
	sh_ex_sigmaxx0[index_blk_ex(threadIdx.x-5,threadIdx.y-5)] = g_ex_sigmaxx0[index3d_ex(iz-5,ix-5,it+1)];
	sh_ex_sigmazz0[index_blk_ex(threadIdx.x-5,threadIdx.y-5)] = g_ex_sigmazz0[index3d_ex(iz-5,ix-5,it+1)];
	sh_ex_sigmaxz0[index_blk_ex(threadIdx.x-5,threadIdx.y-5)] = g_ex_sigmaxz0[index3d_ex(iz-5,ix-5,it+1)];
	}

	if(threadIdx.x >= 5+TZ && threadIdx.y >= 5+TX){
	sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x+5,threadIdx.y+5)] = g_ex_aux_m2m3_c[index_ex(iz+5,ix+5)];
	sh_ex_aux_m2_c[index_blk_ex(threadIdx.x+5,threadIdx.y+5)] = g_ex_aux_m2_c[index_ex(iz+5,ix+5)];
	sh_ex_aux_m3_c[index_blk_ex(threadIdx.x+5,threadIdx.y+5)] = g_ex_aux_m3_c[index_ex(iz+5,ix+5)];
	sh_ex_m1_x[index_blk_ex(threadIdx.x+5,threadIdx.y+5)] = g_ex_m1_x[index_ex(iz+5,ix+5)];
	sh_ex_m1_z[index_blk_ex(threadIdx.x+5,threadIdx.y+5)] = g_ex_m1_z[index_ex(iz+5,ix+5)];

	sh_ex_Vx0[index_blk_ex(threadIdx.x+5,threadIdx.y+5)] = g_ex_Vx0[index3d_ex(iz+5,ix+5,it+1)];
	sh_ex_Vz0[index_blk_ex(threadIdx.x+5,threadIdx.y+5)] = g_ex_Vz0[index3d_ex(iz+5,ix+5,it+1)];
	sh_ex_sigmaxx0[index_blk_ex(threadIdx.x+5,threadIdx.y+5)] = g_ex_sigmaxx0[index3d_ex(iz+5,ix+5,it+1)];
	sh_ex_sigmazz0[index_blk_ex(threadIdx.x+5,threadIdx.y+5)] = g_ex_sigmazz0[index3d_ex(iz+5,ix+5,it+1)];
	sh_ex_sigmaxz0[index_blk_ex(threadIdx.x+5,threadIdx.y+5)] = g_ex_sigmaxz0[index3d_ex(iz+5,ix+5,it+1)];
	}


	if(threadIdx.x >= TZ+5 && threadIdx.y <5){
	sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x+5,threadIdx.y-5)] = g_ex_aux_m2m3_c[index_ex(iz+5,ix-5)];
	sh_ex_aux_m2_c[index_blk_ex(threadIdx.x+5,threadIdx.y-5)] = g_ex_aux_m2_c[index_ex(iz+5,ix-5)];
	sh_ex_aux_m3_c[index_blk_ex(threadIdx.x+5,threadIdx.y-5)] = g_ex_aux_m3_c[index_ex(iz+5,ix-5)];
	sh_ex_m1_x[index_blk_ex(threadIdx.x+5,threadIdx.y-5)] = g_ex_m1_x[index_ex(iz+5,ix-5)];
	sh_ex_m1_z[index_blk_ex(threadIdx.x+5,threadIdx.y-5)] = g_ex_m1_z[index_ex(iz+5,ix-5)];
	
	sh_ex_Vx0[index_blk_ex(threadIdx.x+5,threadIdx.y-5)] = g_ex_Vx0[index3d_ex(iz+5,ix-5,it+1)];
	sh_ex_Vz0[index_blk_ex(threadIdx.x+5,threadIdx.y-5)] = g_ex_Vz0[index3d_ex(iz+5,ix-5,it+1)];
	sh_ex_sigmaxx0[index_blk_ex(threadIdx.x+5,threadIdx.y-5)] = g_ex_sigmaxx0[index3d_ex(iz+5,ix-5,it+1)];
	sh_ex_sigmazz0[index_blk_ex(threadIdx.x+5,threadIdx.y-5)] = g_ex_sigmazz0[index3d_ex(iz+5,ix-5,it+1)];
	sh_ex_sigmaxz0[index_blk_ex(threadIdx.x+5,threadIdx.y-5)] = g_ex_sigmaxz0[index3d_ex(iz+5,ix-5,it+1)];
	}


	if(threadIdx.x <5 && threadIdx.y >= TX-5){
	sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x-5,threadIdx.y+5)] = g_ex_aux_m2m3_c[index_ex(iz-5,ix+5)];
	sh_ex_aux_m2_c[index_blk_ex(threadIdx.x-5,threadIdx.y+5)] = g_ex_aux_m2_c[index_ex(iz-5,ix+5)];
	sh_ex_aux_m3_c[index_blk_ex(threadIdx.x-5,threadIdx.y+5)] = g_ex_aux_m3_c[index_ex(iz-5,ix+5)];
	sh_ex_m1_x[index_blk_ex(threadIdx.x-5,threadIdx.y+5)] = g_ex_m1_x[index_ex(iz-5,ix+5)];
	sh_ex_m1_z[index_blk_ex(threadIdx.x-5,threadIdx.y+5)] = g_ex_m1_z[index_ex(iz-5,ix+5)];

	sh_ex_Vx0[index_blk_ex(threadIdx.x-5,threadIdx.y+5)] = g_ex_Vx0[index3d_ex(iz-5,ix+5,it+1)];
	sh_ex_Vz0[index_blk_ex(threadIdx.x-5,threadIdx.y+5)] = g_ex_Vz0[index3d_ex(iz-5,ix+5,it+1)];
	sh_ex_sigmaxx0[index_blk_ex(threadIdx.x-5,threadIdx.y+5)] = g_ex_sigmaxx0[index3d_ex(iz-5,ix+5,it+1)];
	sh_ex_sigmazz0[index_blk_ex(threadIdx.x-5,threadIdx.y+5)] = g_ex_sigmazz0[index3d_ex(iz-5,ix+5,it+1)];
	sh_ex_sigmaxz0[index_blk_ex(threadIdx.x-5,threadIdx.y+5)] = g_ex_sigmaxz0[index3d_ex(iz-5,ix+5,it+1)];
	}



	__syncthreads();

              g_ex_Vx0[index3d_ex(iz,ix  ,it)] = g_ex_Vx0[index3d_ex(iz,ix  ,it)]	+ g_ex_Vx0[index3d_ex(iz, ix, it+2)]
									+ sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y-5)]*c1*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y-5)]							
							 		+ sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y-4)]*c2*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y-4)]		
									+ sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y-3)]*c3*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y-3)]	
									+ sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y-2)]*c4*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y-2)]	
									+ sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y-1)]*c5*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y-1)]	
									- sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y)]  *c5*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y)]	
									- sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y+1)]*c4*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y+1)]	
									- sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y+2)]*c3*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y+2)]	
									- sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y+3)]*c2*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y+3)]	
									- sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y+4)]*c1*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y+4)]


									+ sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y-5)]*c1*sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y-5)]							
							 		+ sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y-4)]*c2*sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y-4)]		
									+ sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y-3)]*c3*sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y-3)]	
									+ sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y-2)]*c4*sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y-2)]	
									+ sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y-1)]*c5*sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y-1)]	
									- sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y)]  *c5*sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y)]	
									- sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y+1)]*c4*sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y+1)]	
									- sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y+2)]*c3*sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y+2)]	
									- sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y+3)]*c2*sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y+3)]	
									- sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y+4)]*c1*sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y+4)]	
	


									+ sh_ex_aux_m3_c[index_blk_ex(threadIdx.x-4,threadIdx.y)]*c1*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x-4,threadIdx.y)]		
									+ sh_ex_aux_m3_c[index_blk_ex(threadIdx.x-3,threadIdx.y)]*c2*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x-3,threadIdx.y)]	
									+ sh_ex_aux_m3_c[index_blk_ex(threadIdx.x-2,threadIdx.y)]*c3*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x-2,threadIdx.y)]	
									+ sh_ex_aux_m3_c[index_blk_ex(threadIdx.x-1,threadIdx.y)]*c4*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x-1,threadIdx.y)]	
									+ sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,  threadIdx.y)]  *c5*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y)]	
									- sh_ex_aux_m3_c[index_blk_ex(threadIdx.x+1,threadIdx.y)]*c5*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x+1,threadIdx.y)]	
									- sh_ex_aux_m3_c[index_blk_ex(threadIdx.x+2,threadIdx.y)]*c4*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x+2,threadIdx.y)]	
									- sh_ex_aux_m3_c[index_blk_ex(threadIdx.x+3,threadIdx.y)]*c3*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x+3,threadIdx.y)]	
									- sh_ex_aux_m3_c[index_blk_ex(threadIdx.x+4,threadIdx.y)]*c2*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x+4,threadIdx.y)]	
									- sh_ex_aux_m3_c[index_blk_ex(threadIdx.x+5,threadIdx.y)]*c1*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x+5,threadIdx.y)]	;						

 
     __syncthreads();

            g_ex_Vz0[index3d_ex(iz,ix  ,it)] = g_ex_Vz0[index3d_ex(iz,ix,  it)]  	+ g_ex_Vz0[index3d_ex(iz,ix  ,it+2)] 
	     								+ sh_ex_aux_m2_c[index_blk_ex(threadIdx.x-5,threadIdx.y)]*c1*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x-5,threadIdx.y)]							
	     						 		+ sh_ex_aux_m2_c[index_blk_ex(threadIdx.x-4,threadIdx.y)]*c2*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x-4,threadIdx.y)]		
	     								+ sh_ex_aux_m2_c[index_blk_ex(threadIdx.x-3,threadIdx.y)]*c3*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x-3,threadIdx.y)]	
	     								+ sh_ex_aux_m2_c[index_blk_ex(threadIdx.x-2,threadIdx.y)]*c4*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x-2,threadIdx.y)]	
	     								+ sh_ex_aux_m2_c[index_blk_ex(threadIdx.x-1,threadIdx.y)]*c5*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x-1,threadIdx.y)]	
	     								- sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,  threadIdx.y)]  *c5*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y)]	
	     								- sh_ex_aux_m2_c[index_blk_ex(threadIdx.x+1,threadIdx.y)]*c4*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x+1,threadIdx.y)]	
	     								- sh_ex_aux_m2_c[index_blk_ex(threadIdx.x+2,threadIdx.y)]*c3*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x+2,threadIdx.y)]	
	     								- sh_ex_aux_m2_c[index_blk_ex(threadIdx.x+3,threadIdx.y)]*c2*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x+3,threadIdx.y)]	
	     								- sh_ex_aux_m2_c[index_blk_ex(threadIdx.x+4,threadIdx.y)]*c1*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x+4,threadIdx.y)]	
	     
	
	             							+ sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x-5,threadIdx.y)]*c1*sh_ex_sigmazz0[index_blk_ex(threadIdx.x-5,threadIdx.y)]							
	     						 		+ sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x-4,threadIdx.y)]*c2*sh_ex_sigmazz0[index_blk_ex(threadIdx.x-4,threadIdx.y)]		
	     								+ sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x-3,threadIdx.y)]*c3*sh_ex_sigmazz0[index_blk_ex(threadIdx.x-3,threadIdx.y)]	
	     								+ sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x-2,threadIdx.y)]*c4*sh_ex_sigmazz0[index_blk_ex(threadIdx.x-2,threadIdx.y)]	
	     								+ sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x-1,threadIdx.y)]*c5*sh_ex_sigmazz0[index_blk_ex(threadIdx.x-1,threadIdx.y)]	
	     								- sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,  threadIdx.y)]  *c5*sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y)]	
	     								- sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x+1,threadIdx.y)]*c4*sh_ex_sigmazz0[index_blk_ex(threadIdx.x+1,threadIdx.y)]	
	     								- sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x+2,threadIdx.y)]*c3*sh_ex_sigmazz0[index_blk_ex(threadIdx.x+2,threadIdx.y)]	
	     								- sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x+3,threadIdx.y)]*c2*sh_ex_sigmazz0[index_blk_ex(threadIdx.x+3,threadIdx.y)]	
	     								- sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x+4,threadIdx.y)]*c1*sh_ex_sigmazz0[index_blk_ex(threadIdx.x+4,threadIdx.y)]	
	     
	     								+ sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y-4)]*c1*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y-4)]		
	     								+ sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y-3)]*c2*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y-3)]	
	     								+ sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y-2)]*c3*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y-2)]	
	     								+ sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y-1)]*c4*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y-1)]	
	     								+ sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y)]  *c5*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y)]	
	     								- sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y+1)]*c5*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y+1)]	
	     								- sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y+2)]*c4*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y+2)]	
	     								- sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y+3)]*c3*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y+3)]	
	     								- sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y+4)]*c2*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y+4)]	
	     								- sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y+5)]*c1*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y+5)]	;							
	


              g_ex_sigmaxx0[index3d_ex(iz,ix  ,it)] = g_ex_sigmaxx0[index3d_ex(iz,ix  ,it)]	+ g_ex_sigmaxx0[index3d_ex(iz,ix  ,it+2)] 
        									+ sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y-4)]*c1*sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y-4)]		
        									+ sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y-3)]*c2*sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y-3)]	
        									+ sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y-2)]*c3*sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y-2)]	
        									+ sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y-1)]*c4*sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y-1)]	
        									+ sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y)]  *c5*sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y)]	
        									- sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y+1)]*c5*sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y+1)]	
        									- sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y+2)]*c4*sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y+2)]	
        									- sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y+3)]*c3*sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y+3)]	
        									- sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y+4)]*c2*sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y+4)]	
        									- sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y+5)]*c1*sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y+5)]	;						
 
    __syncthreads();
             g_ex_sigmazz0[index3d_ex(iz,ix  ,it)] = g_ex_sigmazz0[index3d_ex(iz,ix  ,it)]	+ g_ex_sigmazz0[index3d_ex(iz,ix  ,it+2)] 
										+ sh_ex_m1_z[index_blk_ex(threadIdx.x-4,threadIdx.y)]*c1*sh_ex_Vz0[index_blk_ex(threadIdx.x-4,threadIdx.y)]		
										+ sh_ex_m1_z[index_blk_ex(threadIdx.x-3,threadIdx.y)]*c2*sh_ex_Vz0[index_blk_ex(threadIdx.x-3,threadIdx.y)]	
										+ sh_ex_m1_z[index_blk_ex(threadIdx.x-2,threadIdx.y)]*c3*sh_ex_Vz0[index_blk_ex(threadIdx.x-2,threadIdx.y)]	
										+ sh_ex_m1_z[index_blk_ex(threadIdx.x-1,threadIdx.y)]*c4*sh_ex_Vz0[index_blk_ex(threadIdx.x-1,threadIdx.y)]	
										+ sh_ex_m1_z[index_blk_ex(threadIdx.x,  threadIdx.y)]  *c5*sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y)]	
										- sh_ex_m1_z[index_blk_ex(threadIdx.x+1,threadIdx.y)]*c5*sh_ex_Vz0[index_blk_ex(threadIdx.x+1,threadIdx.y)]	
										- sh_ex_m1_z[index_blk_ex(threadIdx.x+2,threadIdx.y)]*c4*sh_ex_Vz0[index_blk_ex(threadIdx.x+2,threadIdx.y)]	
										- sh_ex_m1_z[index_blk_ex(threadIdx.x+3,threadIdx.y)]*c3*sh_ex_Vz0[index_blk_ex(threadIdx.x+3,threadIdx.y)]	
										- sh_ex_m1_z[index_blk_ex(threadIdx.x+4,threadIdx.y)]*c2*sh_ex_Vz0[index_blk_ex(threadIdx.x+4,threadIdx.y)]	
										- sh_ex_m1_z[index_blk_ex(threadIdx.x+5,threadIdx.y)]*c1*sh_ex_Vz0[index_blk_ex(threadIdx.x+5,threadIdx.y)]	;						
     __syncthreads();
     g_ex_sigmaxz0[index3d_ex(iz,ix  ,it)] = g_ex_sigmaxz0[index3d_ex(iz,ix  ,it)]	+ g_ex_sigmaxz0[index3d_ex(iz,ix  ,it+2)]	 
										+ sh_ex_m1_x[index_blk_ex(threadIdx.x-5,threadIdx.y)]*c1*sh_ex_Vx0[index_blk_ex(threadIdx.x-5,threadIdx.y)]							
							 			+ sh_ex_m1_x[index_blk_ex(threadIdx.x-4,threadIdx.y)]*c2*sh_ex_Vx0[index_blk_ex(threadIdx.x-4,threadIdx.y)]		
										+ sh_ex_m1_x[index_blk_ex(threadIdx.x-3,threadIdx.y)]*c3*sh_ex_Vx0[index_blk_ex(threadIdx.x-3,threadIdx.y)]	
										+ sh_ex_m1_x[index_blk_ex(threadIdx.x-2,threadIdx.y)]*c4*sh_ex_Vx0[index_blk_ex(threadIdx.x-2,threadIdx.y)]	
										+ sh_ex_m1_x[index_blk_ex(threadIdx.x-1,threadIdx.y)]*c5*sh_ex_Vx0[index_blk_ex(threadIdx.x-1,threadIdx.y)]	
										- sh_ex_m1_x[index_blk_ex(threadIdx.x,  threadIdx.y)]  *c5*sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y)]	
										- sh_ex_m1_x[index_blk_ex(threadIdx.x+1,threadIdx.y)]*c4*sh_ex_Vx0[index_blk_ex(threadIdx.x+1,threadIdx.y)]	
										- sh_ex_m1_x[index_blk_ex(threadIdx.x+2,threadIdx.y)]*c3*sh_ex_Vx0[index_blk_ex(threadIdx.x+2,threadIdx.y)]	
										- sh_ex_m1_x[index_blk_ex(threadIdx.x+3,threadIdx.y)]*c2*sh_ex_Vx0[index_blk_ex(threadIdx.x+3,threadIdx.y)]	
										- sh_ex_m1_x[index_blk_ex(threadIdx.x+4,threadIdx.y)]*c1*sh_ex_Vx0[index_blk_ex(threadIdx.x+4,threadIdx.y)]	//;
	
        
										+ sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y-5)]*c1*sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y-5)]							
							 			+ sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y-4)]*c2*sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y-4)]		
										+ sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y-3)]*c3*sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y-3)]	
										+ sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y-2)]*c4*sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y-2)]	
										+ sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y-1)]*c5*sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y-1)]	
										- sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y)]  *c5*sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y)]	
										- sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y+1)]*c4*sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y+1)]	
										- sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y+2)]*c3*sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y+2)]	
										- sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y+3)]*c2*sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y+3)]	
										- sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y+4)]*c1*sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y+4)]	;
		
	__syncthreads();


	}


__global__ void rtm_gpu_kernel_l1(int it,int nt, int nz, int nx,
        float * g_ex_Vx0, float * g_ex_Vz0, float * g_ex_sigmaxx0, float * g_ex_sigmazz0, float * g_ex_sigmaxz0, //(nz, nx, nt)
        float * g_ex_m1_x,float * g_ex_m1_z,float * g_ex_aux_m2_c, float * g_ex_aux_m3_c, float * g_ex_aux_m2m3_c)//(nz+10,	nx+10)
{

	float c1=35.0/294912.0,c2=-405.0/229376.0,c3=567.0/40960.0,c4=-735.0/8192.0,c5=19845.0/16384.0;

	//GPU thread index
	int iz, ix;
	iz = blockIdx.x*blockDim.x + threadIdx.x;
	ix = blockIdx.y*blockDim.y + threadIdx.y;
	//gt = it;
 	
              g_ex_Vx0[index3d_ex(iz,ix  ,it)] = g_ex_Vx0[index3d_ex(iz,ix  ,it)]	+ g_ex_Vx0[index3d_ex(iz, ix, it+2)]
									+ g_ex_aux_m2m3_c[index_ex(iz,ix-5)]*c1*g_ex_sigmaxx0[index3d_ex(iz,ix-5,it+1)]							
							 		+ g_ex_aux_m2m3_c[index_ex(iz,ix-4)]*c2*g_ex_sigmaxx0[index3d_ex(iz,ix-4,it+1)]		
									+ g_ex_aux_m2m3_c[index_ex(iz,ix-3)]*c3*g_ex_sigmaxx0[index3d_ex(iz,ix-3,it+1)]	
									+ g_ex_aux_m2m3_c[index_ex(iz,ix-2)]*c4*g_ex_sigmaxx0[index3d_ex(iz,ix-2,it+1)]	
									+ g_ex_aux_m2m3_c[index_ex(iz,ix-1)]*c5*g_ex_sigmaxx0[index3d_ex(iz,ix-1,it+1)]	
									- g_ex_aux_m2m3_c[index_ex(iz,ix)]  *c5*g_ex_sigmaxx0[index3d_ex(iz,ix,it+1)]	
									- g_ex_aux_m2m3_c[index_ex(iz,ix+1)]*c4*g_ex_sigmaxx0[index3d_ex(iz,ix+1,it+1)]	
									- g_ex_aux_m2m3_c[index_ex(iz,ix+2)]*c3*g_ex_sigmaxx0[index3d_ex(iz,ix+2,it+1)]	
									- g_ex_aux_m2m3_c[index_ex(iz,ix+3)]*c2*g_ex_sigmaxx0[index3d_ex(iz,ix+3,it+1)]	
									- g_ex_aux_m2m3_c[index_ex(iz,ix+4)]*c1*g_ex_sigmaxx0[index3d_ex(iz,ix+4,it+1)]


									+ g_ex_aux_m2_c[index_ex(iz,ix-5)]*c1*g_ex_sigmazz0[index3d_ex(iz,ix-5,it+1)]							
							 		+ g_ex_aux_m2_c[index_ex(iz,ix-4)]*c2*g_ex_sigmazz0[index3d_ex(iz,ix-4,it+1)]		
									+ g_ex_aux_m2_c[index_ex(iz,ix-3)]*c3*g_ex_sigmazz0[index3d_ex(iz,ix-3,it+1)]	
									+ g_ex_aux_m2_c[index_ex(iz,ix-2)]*c4*g_ex_sigmazz0[index3d_ex(iz,ix-2,it+1)]	
									+ g_ex_aux_m2_c[index_ex(iz,ix-1)]*c5*g_ex_sigmazz0[index3d_ex(iz,ix-1,it+1)]	
									- g_ex_aux_m2_c[index_ex(iz,ix)]  *c5*g_ex_sigmazz0[index3d_ex(iz,ix,it+1)]	
									- g_ex_aux_m2_c[index_ex(iz,ix+1)]*c4*g_ex_sigmazz0[index3d_ex(iz,ix+1,it+1)]	
									- g_ex_aux_m2_c[index_ex(iz,ix+2)]*c3*g_ex_sigmazz0[index3d_ex(iz,ix+2,it+1)]	
									- g_ex_aux_m2_c[index_ex(iz,ix+3)]*c2*g_ex_sigmazz0[index3d_ex(iz,ix+3,it+1)]	
									- g_ex_aux_m2_c[index_ex(iz,ix+4)]*c1*g_ex_sigmazz0[index3d_ex(iz,ix+4,it+1)]	
	


									+ g_ex_aux_m3_c[index_ex(iz-4,ix)]*c1*g_ex_sigmaxz0[index3d_ex(iz-4,ix,it+1)]		
									+ g_ex_aux_m3_c[index_ex(iz-3,ix)]*c2*g_ex_sigmaxz0[index3d_ex(iz-3,ix,it+1)]	
									+ g_ex_aux_m3_c[index_ex(iz-2,ix)]*c3*g_ex_sigmaxz0[index3d_ex(iz-2,ix,it+1)]	
									+ g_ex_aux_m3_c[index_ex(iz-1,ix)]*c4*g_ex_sigmaxz0[index3d_ex(iz-1,ix,it+1)]	
									+ g_ex_aux_m3_c[index_ex(iz,ix)]  *c5*g_ex_sigmaxz0[index3d_ex(iz,ix,it+1)]	
									- g_ex_aux_m3_c[index_ex(iz+1,ix)]*c5*g_ex_sigmaxz0[index3d_ex(iz+1,ix,it+1)]	
									- g_ex_aux_m3_c[index_ex(iz+2,ix)]*c4*g_ex_sigmaxz0[index3d_ex(iz+2,ix,it+1)]	
									- g_ex_aux_m3_c[index_ex(iz+3,ix)]*c3*g_ex_sigmaxz0[index3d_ex(iz+3,ix,it+1)]	
									- g_ex_aux_m3_c[index_ex(iz+4,ix)]*c2*g_ex_sigmaxz0[index3d_ex(iz+4,ix,it+1)]	
									- g_ex_aux_m3_c[index_ex(iz+5,ix)]*c1*g_ex_sigmaxz0[index3d_ex(iz+5,ix,it+1)]	;						

 

            g_ex_Vz0[index3d_ex(iz,ix  ,it)] = g_ex_Vz0[index3d_ex(iz,ix  ,it)]  	+ g_ex_Vz0[index3d_ex(iz,ix  ,it+2)] 
	     								+ g_ex_aux_m2_c[index_ex(iz-5,ix)]*c1*g_ex_sigmaxx0[index3d_ex(iz-5,ix,it+1)]							
	     						 		+ g_ex_aux_m2_c[index_ex(iz-4,ix)]*c2*g_ex_sigmaxx0[index3d_ex(iz-4,ix,it+1)]		
	     								+ g_ex_aux_m2_c[index_ex(iz-3,ix)]*c3*g_ex_sigmaxx0[index3d_ex(iz-3,ix,it+1)]	
	     								+ g_ex_aux_m2_c[index_ex(iz-2,ix)]*c4*g_ex_sigmaxx0[index3d_ex(iz-2,ix,it+1)]	
	     								+ g_ex_aux_m2_c[index_ex(iz-1,ix)]*c5*g_ex_sigmaxx0[index3d_ex(iz-1,ix,it+1)]	
	     								- g_ex_aux_m2_c[index_ex(iz,ix)]  *c5*g_ex_sigmaxx0[index3d_ex(iz,ix,it+1)]	
	     								- g_ex_aux_m2_c[index_ex(iz+1,ix)]*c4*g_ex_sigmaxx0[index3d_ex(iz+1,ix,it+1)]	
	     								- g_ex_aux_m2_c[index_ex(iz+2,ix)]*c3*g_ex_sigmaxx0[index3d_ex(iz+2,ix,it+1)]	
	     								- g_ex_aux_m2_c[index_ex(iz+3,ix)]*c2*g_ex_sigmaxx0[index3d_ex(iz+3,ix,it+1)]	
	     								- g_ex_aux_m2_c[index_ex(iz+4,ix)]*c1*g_ex_sigmaxx0[index3d_ex(iz+4,ix,it+1)]	
	     
	
	             							+ g_ex_aux_m2m3_c[index_ex(iz-5,ix)]*c1*g_ex_sigmazz0[index3d_ex(iz-5,ix,it+1)]							
	     						 		+ g_ex_aux_m2m3_c[index_ex(iz-4,ix)]*c2*g_ex_sigmazz0[index3d_ex(iz-4,ix,it+1)]		
	     								+ g_ex_aux_m2m3_c[index_ex(iz-3,ix)]*c3*g_ex_sigmazz0[index3d_ex(iz-3,ix,it+1)]	
	     								+ g_ex_aux_m2m3_c[index_ex(iz-2,ix)]*c4*g_ex_sigmazz0[index3d_ex(iz-2,ix,it+1)]	
	     								+ g_ex_aux_m2m3_c[index_ex(iz-1,ix)]*c5*g_ex_sigmazz0[index3d_ex(iz-1,ix,it+1)]	
	     								- g_ex_aux_m2m3_c[index_ex(iz,ix)]  *c5*g_ex_sigmazz0[index3d_ex(iz,ix,it+1)]	
	     								- g_ex_aux_m2m3_c[index_ex(iz+1,ix)]*c4*g_ex_sigmazz0[index3d_ex(iz+1,ix,it+1)]	
	     								- g_ex_aux_m2m3_c[index_ex(iz+2,ix)]*c3*g_ex_sigmazz0[index3d_ex(iz+2,ix,it+1)]	
	     								- g_ex_aux_m2m3_c[index_ex(iz+3,ix)]*c2*g_ex_sigmazz0[index3d_ex(iz+3,ix,it+1)]	
	     								- g_ex_aux_m2m3_c[index_ex(iz+4,ix)]*c1*g_ex_sigmazz0[index3d_ex(iz+4,ix,it+1)]	
	     
	     								+ g_ex_aux_m3_c[index_ex(iz,ix-4)]*c1*g_ex_sigmaxz0[index3d_ex(iz,ix-4,it+1)]		
	     								+ g_ex_aux_m3_c[index_ex(iz,ix-3)]*c2*g_ex_sigmaxz0[index3d_ex(iz,ix-3,it+1)]	
	     								+ g_ex_aux_m3_c[index_ex(iz,ix-2)]*c3*g_ex_sigmaxz0[index3d_ex(iz,ix-2,it+1)]	
	     								+ g_ex_aux_m3_c[index_ex(iz,ix-1)]*c4*g_ex_sigmaxz0[index3d_ex(iz,ix-1,it+1)]	
	     								+ g_ex_aux_m3_c[index_ex(iz,ix)]  *c5*g_ex_sigmaxz0[index3d_ex(iz,ix,it+1)]	
	     								- g_ex_aux_m3_c[index_ex(iz,ix+1)]*c5*g_ex_sigmaxz0[index3d_ex(iz,ix+1,it+1)]	
	     								- g_ex_aux_m3_c[index_ex(iz,ix+2)]*c4*g_ex_sigmaxz0[index3d_ex(iz,ix+2,it+1)]	
	     								- g_ex_aux_m3_c[index_ex(iz,ix+3)]*c3*g_ex_sigmaxz0[index3d_ex(iz,ix+3,it+1)]	
	     								- g_ex_aux_m3_c[index_ex(iz,ix+4)]*c2*g_ex_sigmaxz0[index3d_ex(iz,ix+4,it+1)]	
	     								- g_ex_aux_m3_c[index_ex(iz,ix+5)]*c1*g_ex_sigmaxz0[index3d_ex(iz,ix+5,it+1)]	;							
	


              g_ex_sigmaxx0[index3d_ex(iz,ix  ,it)] = g_ex_sigmaxx0[index3d_ex(iz,ix  ,it)]	+ g_ex_sigmaxx0[index3d_ex(iz,ix  ,it+2)] 
        									+ g_ex_m1_x[index_ex(iz,ix-4)]*c1*g_ex_Vx0[index3d_ex(iz,ix-4,it+1)]		
        									+ g_ex_m1_x[index_ex(iz,ix-3)]*c2*g_ex_Vx0[index3d_ex(iz,ix-3,it+1)]	
        									+ g_ex_m1_x[index_ex(iz,ix-2)]*c3*g_ex_Vx0[index3d_ex(iz,ix-2,it+1)]	
        									+ g_ex_m1_x[index_ex(iz,ix-1)]*c4*g_ex_Vx0[index3d_ex(iz,ix-1,it+1)]	
        									+ g_ex_m1_x[index_ex(iz,ix)]  *c5*g_ex_Vx0[index3d_ex(iz,ix,it+1)]	
        									- g_ex_m1_x[index_ex(iz,ix+1)]*c5*g_ex_Vx0[index3d_ex(iz,ix+1,it+1)]	
        									- g_ex_m1_x[index_ex(iz,ix+2)]*c4*g_ex_Vx0[index3d_ex(iz,ix+2,it+1)]	
        									- g_ex_m1_x[index_ex(iz,ix+3)]*c3*g_ex_Vx0[index3d_ex(iz,ix+3,it+1)]	
        									- g_ex_m1_x[index_ex(iz,ix+4)]*c2*g_ex_Vx0[index3d_ex(iz,ix+4,it+1)]	
        									- g_ex_m1_x[index_ex(iz,ix+5)]*c1*g_ex_Vx0[index3d_ex(iz,ix+5,it+1)]	;						
 
             g_ex_sigmazz0[index3d_ex(iz,ix  ,it)] = g_ex_sigmazz0[index3d_ex(iz,ix  ,it)]	+ g_ex_sigmazz0[index3d_ex(iz,ix  ,it+2)] 
										+ g_ex_m1_z[index_ex(iz-4,ix)]*c1*g_ex_Vz0[index3d_ex(iz-4,ix,it+1)]		
										+ g_ex_m1_z[index_ex(iz-3,ix)]*c2*g_ex_Vz0[index3d_ex(iz-3,ix,it+1)]	
										+ g_ex_m1_z[index_ex(iz-2,ix)]*c3*g_ex_Vz0[index3d_ex(iz-2,ix,it+1)]	
										+ g_ex_m1_z[index_ex(iz-1,ix)]*c4*g_ex_Vz0[index3d_ex(iz-1,ix,it+1)]	
										+ g_ex_m1_z[index_ex(iz,ix)]  *c5*g_ex_Vz0[index3d_ex(iz,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz+1,ix)]*c5*g_ex_Vz0[index3d_ex(iz+1,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz+2,ix)]*c4*g_ex_Vz0[index3d_ex(iz+2,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz+3,ix)]*c3*g_ex_Vz0[index3d_ex(iz+3,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz+4,ix)]*c2*g_ex_Vz0[index3d_ex(iz+4,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz+5,ix)]*c1*g_ex_Vz0[index3d_ex(iz+5,ix,it+1)]	;						
     
	g_ex_sigmaxz0[index3d_ex(iz,ix  ,it)] = g_ex_sigmaxz0[index3d_ex(iz,ix  ,it)]	+ g_ex_sigmaxz0[index3d_ex(iz,ix  ,it+2)]	 
										+ g_ex_m1_x[index_ex(iz-5,ix)]*c1*g_ex_Vx0[index3d_ex(iz-5,ix,it+1)]							
							 			+ g_ex_m1_x[index_ex(iz-4,ix)]*c2*g_ex_Vx0[index3d_ex(iz-4,ix,it+1)]		
										+ g_ex_m1_x[index_ex(iz-3,ix)]*c3*g_ex_Vx0[index3d_ex(iz-3,ix,it+1)]	
										+ g_ex_m1_x[index_ex(iz-2,ix)]*c4*g_ex_Vx0[index3d_ex(iz-2,ix,it+1)]	
										+ g_ex_m1_x[index_ex(iz-1,ix)]*c5*g_ex_Vx0[index3d_ex(iz-1,ix,it+1)]	
										- g_ex_m1_x[index_ex(iz,ix)]  *c5*g_ex_Vx0[index3d_ex(iz,ix,it+1)]	
										- g_ex_m1_x[index_ex(iz+1,ix)]*c4*g_ex_Vx0[index3d_ex(iz+1,ix,it+1)]	
										- g_ex_m1_x[index_ex(iz+2,ix)]*c3*g_ex_Vx0[index3d_ex(iz+2,ix,it+1)]	
										- g_ex_m1_x[index_ex(iz+3,ix)]*c2*g_ex_Vx0[index3d_ex(iz+3,ix,it+1)]	
										- g_ex_m1_x[index_ex(iz+4,ix)]*c1*g_ex_Vx0[index3d_ex(iz+4,ix,it+1)]	//;
	
        
										+ g_ex_m1_z[index_ex(iz,ix-5)]*c1*g_ex_Vz0[index3d_ex(iz,ix-5,it+1)]							
							 			+ g_ex_m1_z[index_ex(iz,ix-4)]*c2*g_ex_Vz0[index3d_ex(iz,ix-4,it+1)]		
										+ g_ex_m1_z[index_ex(iz,ix-3)]*c3*g_ex_Vz0[index3d_ex(iz,ix-3,it+1)]	
										+ g_ex_m1_z[index_ex(iz,ix-2)]*c4*g_ex_Vz0[index3d_ex(iz,ix-2,it+1)]	
										+ g_ex_m1_z[index_ex(iz,ix-1)]*c5*g_ex_Vz0[index3d_ex(iz,ix-1,it+1)]	
										- g_ex_m1_z[index_ex(iz,ix)]  *c5*g_ex_Vz0[index3d_ex(iz,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz,ix+1)]*c4*g_ex_Vz0[index3d_ex(iz,ix+1,it+1)]	
										- g_ex_m1_z[index_ex(iz,ix+2)]*c3*g_ex_Vz0[index3d_ex(iz,ix+2,it+1)]	
										- g_ex_m1_z[index_ex(iz,ix+3)]*c2*g_ex_Vz0[index3d_ex(iz,ix+3,it+1)]	
										- g_ex_m1_z[index_ex(iz,ix+4)]*c1*g_ex_Vz0[index3d_ex(iz,ix+4,it+1)]	;
		

	}



