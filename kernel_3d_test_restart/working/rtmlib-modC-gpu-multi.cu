#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "gpu.h"
#include "sep3d.h"
#include "seplib.h"




__constant__ float g_source[TS];

__global__ void rtm_gpu_kernel(int ny, int nz, int nx, int gpu_id, int it, 
        float *g_ex_Vy0_now,  float * g_ex_Vx0_now, float * g_ex_Vz0_now, float * g_ex_sigmayy0_now, float *g_ex_sigmaxx0_now, float * g_ex_sigmazz0_now, float * g_ex_sigmaxy0_now, float * g_ex_sigmaxz0_now, float * g_ex_sigmayz0_now,//(nz, nx, nt)
        float *g_ex_Vy0_next,  float * g_ex_Vx0_next, float * g_ex_Vz0_next, float * g_ex_sigmayy0_next, float *g_ex_sigmaxx0_next, float * g_ex_sigmazz0_next, float * g_ex_sigmaxy0_next, float * g_ex_sigmaxz0_next, float * g_ex_sigmayz0_next,//(nz, nx, nt)
        float *g_ex_Vy0_pre,  float * g_ex_Vx0_pre, float * g_ex_Vz0_pre, float * g_ex_sigmayy0_pre, float *g_ex_sigmaxx0_pre, float * g_ex_sigmazz0_pre, float * g_ex_sigmaxy0_pre, float * g_ex_sigmaxz0_pre, float * g_ex_sigmayz0_pre,//(nz, nx, nt)
     	float * g_ex_m1_y,    float * g_ex_m1_x,    float * g_ex_m1_z,   float *  g_ex_m2,  float * g_ex_m3,  float * g_ex_m2m3);//(nz+10,	nx+10)




extern "C" void setup_cuda(int n_gpus){
	int dr;
	int i, j, k;

	for(i=0; i<n_gpus; i++) device[i] = i;

	for(i=0; i<n_gpus; i++) {
		hipDeviceSynchronize();
		
		hipSetDevice(device[i]);
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, device[i]);
		hipDriverGetVersion(&dr);

		//Enable P2P memcopies between GPUs
		if(n_gpus >1){
			for(j=0;j<n_gpus; j++){
				if(i == j) continue;
				int peer_access_available = 0;
				hipDeviceCanAccessPeer(&peer_access_available, device[i], device[j]);
				if(peer_access_available)
					hipDeviceEnablePeerAccess(device[j], 0);
			}
		}
	}

}


__global__ void add_source_kernel(int ny, int nz, int nx, int source_iy, int source_iz, int source_ix, int it, float * g_ex_sigmayy0_now, float *g_ex_sigmaxx0_now, float * g_ex_sigmazz0_now)//(ny+10, nx+10, nz+10)
{
	float dt = 0.4*(4.0)/(sqrt(2.0)*2000.0);

	int iz, ix, iy;
	iz = blockIdx.x*blockDim.x + threadIdx.x;
	ix = blockIdx.y*blockDim.y + threadIdx.y;
	iy = blockIdx.z*blockDim.z + threadIdx.z;

	//add_source
	if(iy==source_iy && ix==source_ix && iz==source_iz){
		g_ex_sigmayy0_now[n3d_index_ex(iz,ix  ,iy)] += dt*g_source[it];
		g_ex_sigmaxx0_now[n3d_index_ex(iz,ix  ,iy)] += dt*g_source[it];
		g_ex_sigmazz0_now[n3d_index_ex(iz,ix  ,iy)] += dt*g_source[it];
	}

}


__global__ void rtm_gpu_kernel(int ny, int nz, int nx,//int source_iy, int source_ix, int source_iz,
        float *g_ex_Vy0_now,  float * g_ex_Vx0_now, float * g_ex_Vz0_now, float * g_ex_sigmayy0_now, float *g_ex_sigmaxx0_now, float * g_ex_sigmazz0_now, float * g_ex_sigmaxy0_now, float * g_ex_sigmaxz0_now, float * g_ex_sigmayz0_now,//(ny+10, nx+10, nz+10)
        float *g_ex_Vy0_next,  float * g_ex_Vx0_next, float * g_ex_Vz0_next, float * g_ex_sigmayy0_next, float *g_ex_sigmaxx0_next, float * g_ex_sigmazz0_next, float * g_ex_sigmaxy0_next, float * g_ex_sigmaxz0_next, float * g_ex_sigmayz0_next,//(ny+10,nx+10,nz+10)
        float *g_ex_Vy0_pre,  float * g_ex_Vx0_pre, float * g_ex_Vz0_pre, float * g_ex_sigmayy0_pre, float *g_ex_sigmaxx0_pre, float * g_ex_sigmazz0_pre, float * g_ex_sigmaxy0_pre, float * g_ex_sigmaxz0_pre, float * g_ex_sigmayz0_pre,//(ny+10,nx+10,nz+10) 
     	float * g_ex_m1_y,    float * g_ex_m1_x,    float * g_ex_m1_z,  float * g_ex_m2, float * g_ex_m3,  float * g_ex_m2m3)//(ny+10,nx+10,nz+10) 
{

	float c1=35.0/294912.0,c2=-405.0/229376.0,c3=567.0/40960.0,c4=-735.0/8192.0,c5=19845.0/16384.0;

	//GPU thread index
	int iz, ix, iy;
	iz = blockIdx.x*blockDim.x + threadIdx.x;
	ix = blockIdx.y*blockDim.y + threadIdx.y;
	iy = blockIdx.z*blockDim.z + threadIdx.z;

//	__shared__ float sh_g_ex_Vx0_now[(TX+10)*(TZ+10)*(TY+10)];
//	__shared__ float sh_g_ex_Vy0_now[(TX+10)*(TZ+10)*(TY+10)];
//	__shared__ float sh_g_ex_Vz0_now[(TX+10)*(TZ+10)*(TY+10)];
//	__shared__ float sh_g_ex_sigmaxx0_now[(TX+10)*(TZ+10)*(TY+10)];
//	__shared__ float sh_g_ex_sigmayy0_now[(TX+10)*(TZ+10)*(TY+10)];
//	__shared__ float sh_g_ex_sigmazz0_now[(TX+10)*(TZ+10)*(TY+10)];
//	__shared__ float sh_g_ex_sigmaxy0_now[(TX+10)*(TZ+10)*(TY+10)];
//	__shared__ float sh_g_ex_sigmayz0_now[(TX+10)*(TZ+10)*(TY+10)];
//	__shared__ float sh_g_ex_sigmaxz0_now[(TX+10)*(TZ+10)*(TY+10)];
//
//
//	sh_g_ex_Vx0_now[n3d_index_blk_ex(threadIdx.x, threadIdx.y, threadIdx.z)] = g_ex_Vx0_pre[n3d_index_ex(iz, ix, iy)];
//	sh_g_ex_Vy0_now[n3d_index_blk_ex(threadIdx.x, threadIdx.y, threadIdx.z)] = g_ex_Vy0_pre[n3d_index_ex(iz, ix, iy)];
//	sh_g_ex_Vz0_now[n3d_index_blk_ex(threadIdx.x, threadIdx.y, threadIdx.z)] = g_ex_Vz0_pre[n3d_index_ex(iz, ix, iy)];
//	sh_g_ex_sigmaxx0_now[n3d_index_blk_ex(threadIdx.x, threadIdx.y, threadIdx.z)] = g_ex_sigmaxx0_pre[n3d_index_ex(iz, ix, iy)];
//	sh_g_ex_sigmayy0_now[n3d_index_blk_ex(threadIdx.x, threadIdx.y, threadIdx.z)] = g_ex_sigmayy0_pre[n3d_index_ex(iz, ix, iy)];
//	sh_g_ex_sigmazz0_now[n3d_index_blk_ex(threadIdx.x, threadIdx.y, threadIdx.z)] = g_ex_sigmazz0_pre[n3d_index_ex(iz, ix, iy)];
//	sh_g_ex_sigmaxy0_now[n3d_index_blk_ex(threadIdx.x, threadIdx.y, threadIdx.z)] = g_ex_sigmaxy0_pre[n3d_index_ex(iz, ix, iy)];
//	sh_g_ex_sigmaxz0_now[n3d_index_blk_ex(threadIdx.x, threadIdx.y, threadIdx.z)] = g_ex_sigmaxz0_pre[n3d_index_ex(iz, ix, iy)];
//	sh_g_ex_sigmayz0_now[n3d_index_blk_ex(threadIdx.x, threadIdx.y, threadIdx.z)] = g_ex_sigmayz0_pre[n3d_index_ex(iz, ix, iy)];
 

       	g_ex_Vx0_pre[n3d_index_ex(iz,ix  ,iy)] = /*g_ex_Vx0_pre[n3d_index_ex(iz,ix  ,iy)]*/	+ g_ex_Vx0_next[n3d_index_ex(iz, ix, iy)]	

									+ g_ex_m2m3[n3d_index_ex(iz,ix-5, iy)]*c1*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix-5,iy)]							
							 		+ g_ex_m2m3[n3d_index_ex(iz,ix-4, iy)]*c2*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m2m3[n3d_index_ex(iz,ix-3, iy)]*c3*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m2m3[n3d_index_ex(iz,ix-2, iy)]*c4*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m2m3[n3d_index_ex(iz,ix-1, iy)]*c5*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix-1,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix+1, iy)]*c4*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix+2, iy)]*c3*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix+3, iy)]*c2*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix+4, iy)]*c1*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix+4,iy)]
	

									+ g_ex_m2[n3d_index_ex(iz,ix-5, iy)]*c1*g_ex_sigmayy0_now[n3d_index_ex(iz,ix-5,iy)]							
							 		+ g_ex_m2[n3d_index_ex(iz,ix-4, iy)]*c2*g_ex_sigmayy0_now[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m2[n3d_index_ex(iz,ix-3, iy)]*c3*g_ex_sigmayy0_now[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m2[n3d_index_ex(iz,ix-2, iy)]*c4*g_ex_sigmayy0_now[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m2[n3d_index_ex(iz,ix-1, iy)]*c5*g_ex_sigmayy0_now[n3d_index_ex(iz,ix-1,iy)]	
									- g_ex_m2[n3d_index_ex(iz,  ix, iy)]*c5*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+1, iy)]*c4*g_ex_sigmayy0_now[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+2, iy)]*c3*g_ex_sigmayy0_now[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+3, iy)]*c2*g_ex_sigmayy0_now[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+4, iy)]*c1*g_ex_sigmayy0_now[n3d_index_ex(iz,ix+4,iy)]	
	

									+ g_ex_m2[n3d_index_ex(iz,ix-5, iy)]*c1*g_ex_sigmazz0_now[n3d_index_ex(iz,ix-5,iy)]							
							 		+ g_ex_m2[n3d_index_ex(iz,ix-4, iy)]*c2*g_ex_sigmazz0_now[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m2[n3d_index_ex(iz,ix-3, iy)]*c3*g_ex_sigmazz0_now[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m2[n3d_index_ex(iz,ix-2, iy)]*c4*g_ex_sigmazz0_now[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m2[n3d_index_ex(iz,ix-1, iy)]*c5*g_ex_sigmazz0_now[n3d_index_ex(iz,ix-1,iy)]	
									- g_ex_m2[n3d_index_ex(iz,  ix, iy)]*c5*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+1, iy)]*c4*g_ex_sigmazz0_now[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+2, iy)]*c3*g_ex_sigmazz0_now[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+3, iy)]*c2*g_ex_sigmazz0_now[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+4, iy)]*c1*g_ex_sigmazz0_now[n3d_index_ex(iz,ix+4,iy)]	
	

									+ g_ex_m3[n3d_index_ex(iz,ix, iy-4)]*c1*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m3[n3d_index_ex(iz,ix, iy-3)]*c2*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy-2)]*c3*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy-1)]*c4*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix,iy-1)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+1)]*c5*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+2)]*c4*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+3)]*c3*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+4)]*c2*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix,iy+4)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+5)]*c1*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix,iy+5)]							
	

									+ g_ex_m3[n3d_index_ex(iz-4,ix, iy)]*c1*g_ex_sigmaxz0_now[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m3[n3d_index_ex(iz-3,ix, iy)]*c2*g_ex_sigmaxz0_now[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m3[n3d_index_ex(iz-2,ix, iy)]*c3*g_ex_sigmaxz0_now[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m3[n3d_index_ex(iz-1,ix, iy)]*c4*g_ex_sigmaxz0_now[n3d_index_ex(iz-1,ix,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,  ix, iy)]*c5*g_ex_sigmaxz0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+1,ix, iy)]*c5*g_ex_sigmaxz0_now[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+2,ix, iy)]*c4*g_ex_sigmaxz0_now[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+3,ix, iy)]*c3*g_ex_sigmaxz0_now[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+4,ix, iy)]*c2*g_ex_sigmaxz0_now[n3d_index_ex(iz+4,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+5,ix, iy)]*c1*g_ex_sigmaxz0_now[n3d_index_ex(iz+5,ix,iy)]	;						
	


         	g_ex_Vy0_pre[n3d_index_ex(iz,ix  ,iy)] = /*g_ex_Vy0_pre[n3d_index_ex(iz,ix  ,iy)]*/	+ g_ex_Vy0_next[n3d_index_ex(iz, ix, iy)]	

									+ g_ex_m2m3[n3d_index_ex(iz,ix, iy-5)]*c1*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy-5)]							
							 		+ g_ex_m2m3[n3d_index_ex(iz,ix, iy-4)]*c2*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m2m3[n3d_index_ex(iz,ix, iy-3)]*c3*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m2m3[n3d_index_ex(iz,ix, iy-2)]*c4*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m2m3[n3d_index_ex(iz,ix, iy-1)]*c5*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy-1)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix, iy+1)]*c4*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix, iy+2)]*c3*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix, iy+3)]*c2*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix, iy+4)]*c1*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy+4)]
	

									+ g_ex_m2[n3d_index_ex(iz,ix, iy-5)]*c1*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy-5)]							
							 		+ g_ex_m2[n3d_index_ex(iz,ix, iy-4)]*c2*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m2[n3d_index_ex(iz,ix, iy-3)]*c3*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m2[n3d_index_ex(iz,ix, iy-2)]*c4*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m2[n3d_index_ex(iz,ix, iy-1)]*c5*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy-1)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+1)]*c4*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+2)]*c3*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+3)]*c2*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+4)]*c1*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy+4)]	
	

									+ g_ex_m2[n3d_index_ex(iz,ix, iy-5)]*c1*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy-5)]							
							 		+ g_ex_m2[n3d_index_ex(iz,ix, iy-4)]*c2*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m2[n3d_index_ex(iz,ix, iy-3)]*c3*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m2[n3d_index_ex(iz,ix, iy-2)]*c4*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m2[n3d_index_ex(iz,ix, iy-1)]*c5*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy-1)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+1)]*c4*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+2)]*c3*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+3)]*c2*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+4)]*c1*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy+4)]	
	

									+ g_ex_m3[n3d_index_ex(iz-4,ix, iy)]*c1*g_ex_sigmayz0_now[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m3[n3d_index_ex(iz-3,ix, iy)]*c2*g_ex_sigmayz0_now[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m3[n3d_index_ex(iz-2,ix, iy)]*c3*g_ex_sigmayz0_now[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m3[n3d_index_ex(iz-1,ix, iy)]*c4*g_ex_sigmayz0_now[n3d_index_ex(iz-1,ix,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmayz0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+1,ix, iy)]*c5*g_ex_sigmayz0_now[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+2,ix, iy)]*c4*g_ex_sigmayz0_now[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+3,ix, iy)]*c3*g_ex_sigmayz0_now[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+4,ix, iy)]*c2*g_ex_sigmayz0_now[n3d_index_ex(iz+4,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+5,ix, iy)]*c1*g_ex_sigmayz0_now[n3d_index_ex(iz+5,ix,iy)]							
	

									+ g_ex_m3[n3d_index_ex(iz,ix-4, iy)]*c1*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m3[n3d_index_ex(iz,ix-3, iy)]*c2*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,ix-2, iy)]*c3*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,ix-1, iy)]*c4*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix-1,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+1, iy)]*c5*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+2, iy)]*c4*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+3, iy)]*c3*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+4, iy)]*c2*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix+4,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+5, iy)]*c1*g_ex_sigmaxy0_now[n3d_index_ex(iz,ix+5,iy)]	;						




         	g_ex_Vz0_pre[n3d_index_ex(iz,ix  ,iy)] = /*g_ex_Vz0_pre[n3d_index_ex(iz,ix  ,iy)]*/	+ g_ex_Vz0_next[n3d_index_ex(iz, ix, iy)]	

									+ g_ex_m2m3[n3d_index_ex(iz-5,ix, iy)]*c1*g_ex_sigmazz0_now[n3d_index_ex(iz-5,ix,iy)]							
							 		+ g_ex_m2m3[n3d_index_ex(iz-4,ix, iy)]*c2*g_ex_sigmazz0_now[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m2m3[n3d_index_ex(iz-3,ix, iy)]*c3*g_ex_sigmazz0_now[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m2m3[n3d_index_ex(iz-2,ix, iy)]*c4*g_ex_sigmazz0_now[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m2m3[n3d_index_ex(iz-1,ix, iy)]*c5*g_ex_sigmazz0_now[n3d_index_ex(iz-1,ix,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmazz0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz+1,ix, iy)]*c4*g_ex_sigmazz0_now[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz+2,ix, iy)]*c3*g_ex_sigmazz0_now[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz+3,ix, iy)]*c2*g_ex_sigmazz0_now[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz+4,ix, iy)]*c1*g_ex_sigmazz0_now[n3d_index_ex(iz+4,ix,iy)]
	

									+ g_ex_m2[n3d_index_ex(iz-5,ix, iy)]*c1*g_ex_sigmaxx0_now[n3d_index_ex(iz-5,ix,iy)]							
							 		+ g_ex_m2[n3d_index_ex(iz-4,ix, iy)]*c2*g_ex_sigmaxx0_now[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m2[n3d_index_ex(iz-3,ix, iy)]*c3*g_ex_sigmaxx0_now[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m2[n3d_index_ex(iz-2,ix, iy)]*c4*g_ex_sigmaxx0_now[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m2[n3d_index_ex(iz-1,ix, iy)]*c5*g_ex_sigmaxx0_now[n3d_index_ex(iz-1,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmaxx0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+1,ix, iy)]*c4*g_ex_sigmaxx0_now[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+2,ix, iy)]*c3*g_ex_sigmaxx0_now[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+3,ix, iy)]*c2*g_ex_sigmaxx0_now[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+4,ix, iy)]*c1*g_ex_sigmaxx0_now[n3d_index_ex(iz+4,ix,iy)]
	

									+ g_ex_m2[n3d_index_ex(iz-5,ix, iy)]*c1*g_ex_sigmayy0_now[n3d_index_ex(iz-5,ix,iy)]							
							 		+ g_ex_m2[n3d_index_ex(iz-4,ix, iy)]*c2*g_ex_sigmayy0_now[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m2[n3d_index_ex(iz-3,ix, iy)]*c3*g_ex_sigmayy0_now[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m2[n3d_index_ex(iz-2,ix, iy)]*c4*g_ex_sigmayy0_now[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m2[n3d_index_ex(iz-1,ix, iy)]*c5*g_ex_sigmayy0_now[n3d_index_ex(iz-1,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmayy0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+1,ix, iy)]*c4*g_ex_sigmayy0_now[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+2,ix, iy)]*c3*g_ex_sigmayy0_now[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+3,ix, iy)]*c2*g_ex_sigmayy0_now[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+4,ix, iy)]*c1*g_ex_sigmayy0_now[n3d_index_ex(iz+4,ix,iy)]
	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy-4)]*c1*g_ex_sigmayz0_now[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m3[n3d_index_ex(iz,ix, iy-3)]*c2*g_ex_sigmayz0_now[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy-2)]*c3*g_ex_sigmayz0_now[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy-1)]*c4*g_ex_sigmayz0_now[n3d_index_ex(iz,ix,iy-1)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmayz0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+1)]*c5*g_ex_sigmayz0_now[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+2)]*c4*g_ex_sigmayz0_now[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+3)]*c3*g_ex_sigmayz0_now[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+4)]*c2*g_ex_sigmayz0_now[n3d_index_ex(iz,ix,iy+4)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+5)]*c1*g_ex_sigmayz0_now[n3d_index_ex(iz,ix,iy+5)]							
	

									+ g_ex_m3[n3d_index_ex(iz,ix-4, iy)]*c1*g_ex_sigmaxz0_now[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m3[n3d_index_ex(iz,ix-3, iy)]*c2*g_ex_sigmaxz0_now[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,ix-2, iy)]*c3*g_ex_sigmaxz0_now[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,ix-1, iy)]*c4*g_ex_sigmaxz0_now[n3d_index_ex(iz,ix-1,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmaxz0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+1, iy)]*c5*g_ex_sigmaxz0_now[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+2, iy)]*c4*g_ex_sigmaxz0_now[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+3, iy)]*c3*g_ex_sigmaxz0_now[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+4, iy)]*c2*g_ex_sigmaxz0_now[n3d_index_ex(iz,ix+4,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+5, iy)]*c1*g_ex_sigmaxz0_now[n3d_index_ex(iz,ix+5,iy)]	;						


		

              g_ex_sigmaxx0_pre[n3d_index_ex(iz,ix  ,iy)] =/* g_ex_sigmaxx0_pre[n3d_index_ex(iz,ix  , iy)]*/	+ g_ex_sigmaxx0_next[n3d_index_ex(iz,ix  , iy)] 
									+ g_ex_m1_x[n3d_index_ex(iz,ix-4, iy)]*c1*g_ex_Vx0_now[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m1_x[n3d_index_ex(iz,ix-3, iy)]*c2*g_ex_Vx0_now[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m1_x[n3d_index_ex(iz,ix-2, iy)]*c3*g_ex_Vx0_now[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m1_x[n3d_index_ex(iz,ix-1, iy)]*c4*g_ex_Vx0_now[n3d_index_ex(iz,ix-1,iy)]	
									+ g_ex_m1_x[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix+1, iy)]*c5*g_ex_Vx0_now[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix+2, iy)]*c4*g_ex_Vx0_now[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix+3, iy)]*c3*g_ex_Vx0_now[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix+4, iy)]*c2*g_ex_Vx0_now[n3d_index_ex(iz,ix+4,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix+5, iy)]*c1*g_ex_Vx0_now[n3d_index_ex(iz,ix+5,iy)]	;						

	    
              g_ex_sigmayy0_pre[n3d_index_ex(iz,ix  ,iy)] = /*g_ex_sigmayy0_pre[n3d_index_ex(iz,ix  , iy)]*/	+ g_ex_sigmayy0_next[n3d_index_ex(iz,ix  , iy)] 
									+ g_ex_m1_y[n3d_index_ex(iz,ix, iy-4)]*c1*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m1_y[n3d_index_ex(iz,ix, iy-3)]*c2*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m1_y[n3d_index_ex(iz,ix, iy-2)]*c3*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m1_y[n3d_index_ex(iz,ix, iy-1)]*c4*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy-1)]	
									+ g_ex_m1_y[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix, iy+1)]*c5*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix, iy+2)]*c4*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix, iy+3)]*c3*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix, iy+4)]*c2*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy+4)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix, iy+5)]*c1*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy+5)]	;		


              g_ex_sigmazz0_pre[n3d_index_ex(iz,ix  ,iy)] = /*g_ex_sigmazz0_pre[n3d_index_ex(iz,ix  , iy)]*/	+ g_ex_sigmazz0_next[n3d_index_ex(iz,ix  , iy)] 
									+ g_ex_m1_z[n3d_index_ex(iz-4,ix, iy)]*c1*g_ex_Vz0_now[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m1_z[n3d_index_ex(iz-3,ix, iy)]*c2*g_ex_Vz0_now[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m1_z[n3d_index_ex(iz-2,ix, iy)]*c3*g_ex_Vz0_now[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m1_z[n3d_index_ex(iz-1,ix, iy)]*c4*g_ex_Vz0_now[n3d_index_ex(iz-1,ix,iy)]	
									+ g_ex_m1_z[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz+1,ix, iy)]*c5*g_ex_Vz0_now[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz+2,ix, iy)]*c4*g_ex_Vz0_now[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz+3,ix, iy)]*c3*g_ex_Vz0_now[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz+4,ix, iy)]*c2*g_ex_Vz0_now[n3d_index_ex(iz+4,ix,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz+5,ix, iy)]*c1*g_ex_Vz0_now[n3d_index_ex(iz+5,ix,iy)]	;		 
	
	


              g_ex_sigmaxy0_pre[n3d_index_ex(iz,ix  ,iy)] = /*g_ex_sigmaxy0_pre[n3d_index_ex(iz,ix  , iy)]*/	+ g_ex_sigmaxy0_next[n3d_index_ex(iz,ix  , iy)] 
									+ g_ex_m1_y[n3d_index_ex(iz,ix-4, iy)]*c1*g_ex_Vy0_now[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m1_y[n3d_index_ex(iz,ix-3, iy)]*c2*g_ex_Vy0_now[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m1_y[n3d_index_ex(iz,ix-2, iy)]*c3*g_ex_Vy0_now[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m1_y[n3d_index_ex(iz,ix-1, iy)]*c4*g_ex_Vy0_now[n3d_index_ex(iz,ix-1,iy)]	
									+ g_ex_m1_y[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix+1, iy)]*c5*g_ex_Vy0_now[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix+2, iy)]*c4*g_ex_Vy0_now[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix+3, iy)]*c3*g_ex_Vy0_now[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix+4, iy)]*c2*g_ex_Vy0_now[n3d_index_ex(iz,ix+4,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix+5, iy)]*c1*g_ex_Vy0_now[n3d_index_ex(iz,ix+5,iy)]	

	    
									+ g_ex_m1_x[n3d_index_ex(iz,ix, iy-4)]*c1*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m1_x[n3d_index_ex(iz,ix, iy-3)]*c2*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m1_x[n3d_index_ex(iz,ix, iy-2)]*c3*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m1_x[n3d_index_ex(iz,ix, iy-1)]*c4*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy-1)]	
									+ g_ex_m1_x[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix, iy+1)]*c5*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix, iy+2)]*c4*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix, iy+3)]*c3*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix, iy+4)]*c2*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy+4)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix, iy+5)]*c1*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy+5)]	;		


              g_ex_sigmaxz0_pre[n3d_index_ex(iz,ix  ,iy)] = /*g_ex_sigmaxz0_pre[n3d_index_ex(iz,ix  , iy)]*/	+ g_ex_sigmaxz0_next[n3d_index_ex(iz,ix  , iy)] 
									+ g_ex_m1_x[n3d_index_ex(iz-4,ix, iy)]*c1*g_ex_Vx0_now[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m1_x[n3d_index_ex(iz-3,ix, iy)]*c2*g_ex_Vx0_now[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m1_x[n3d_index_ex(iz-2,ix, iy)]*c3*g_ex_Vx0_now[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m1_x[n3d_index_ex(iz-1,ix, iy)]*c4*g_ex_Vx0_now[n3d_index_ex(iz-1,ix,iy)]	
									+ g_ex_m1_x[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vx0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz+1,ix, iy)]*c5*g_ex_Vx0_now[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz+2,ix, iy)]*c4*g_ex_Vx0_now[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz+3,ix, iy)]*c3*g_ex_Vx0_now[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz+4,ix, iy)]*c2*g_ex_Vx0_now[n3d_index_ex(iz+4,ix,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz+5,ix, iy)]*c1*g_ex_Vx0_now[n3d_index_ex(iz+5,ix,iy)]	
							
									+ g_ex_m1_z[n3d_index_ex(iz,ix-4, iy)]*c1*g_ex_Vz0_now[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m1_z[n3d_index_ex(iz,ix-3, iy)]*c2*g_ex_Vz0_now[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m1_z[n3d_index_ex(iz,ix-2, iy)]*c3*g_ex_Vz0_now[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m1_z[n3d_index_ex(iz,ix-1, iy)]*c4*g_ex_Vz0_now[n3d_index_ex(iz,ix-1,iy)]	
									+ g_ex_m1_z[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix+1, iy)]*c5*g_ex_Vz0_now[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix+2, iy)]*c4*g_ex_Vz0_now[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix+3, iy)]*c3*g_ex_Vz0_now[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix+4, iy)]*c2*g_ex_Vz0_now[n3d_index_ex(iz,ix+4,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix+5, iy)]*c1*g_ex_Vz0_now[n3d_index_ex(iz,ix+5,iy)]	;						


              g_ex_sigmayz0_pre[n3d_index_ex(iz,ix  ,iy)] = /*g_ex_sigmayz0_pre[n3d_index_ex(iz,ix  , iy)]*/	+ g_ex_sigmayz0_next[n3d_index_ex(iz,ix  , iy)] 
									+ g_ex_m1_y[n3d_index_ex(iz-4,ix, iy)]*c1*g_ex_Vy0_now[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m1_y[n3d_index_ex(iz-3,ix, iy)]*c2*g_ex_Vy0_now[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m1_y[n3d_index_ex(iz-2,ix, iy)]*c3*g_ex_Vy0_now[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m1_y[n3d_index_ex(iz-1,ix, iy)]*c4*g_ex_Vy0_now[n3d_index_ex(iz-1,ix,iy)]	
									+ g_ex_m1_y[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vy0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz+1,ix, iy)]*c5*g_ex_Vy0_now[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz+2,ix, iy)]*c4*g_ex_Vy0_now[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz+3,ix, iy)]*c3*g_ex_Vy0_now[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz+4,ix, iy)]*c2*g_ex_Vy0_now[n3d_index_ex(iz+4,ix,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz+5,ix, iy)]*c1*g_ex_Vy0_now[n3d_index_ex(iz+5,ix,iy)]	
	
									+ g_ex_m1_z[n3d_index_ex(iz,ix, iy-4)]*c1*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m1_z[n3d_index_ex(iz,ix, iy-3)]*c2*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m1_z[n3d_index_ex(iz,ix, iy-2)]*c3*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m1_z[n3d_index_ex(iz,ix, iy-1)]*c4*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy-1)]	
									+ g_ex_m1_z[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix, iy+1)]*c5*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix, iy+2)]*c4*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix, iy+3)]*c3*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix, iy+4)]*c2*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy+4)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix, iy+5)]*c1*g_ex_Vz0_now[n3d_index_ex(iz,ix,iy+5)]	;		

}


extern "C" void rtm_gpu_func(int ny, int nz, int nx, 
        float *ex_Vy0_now,  float * ex_Vx0_now, float * ex_Vz0_now, float * ex_sigmayy0_now, float *ex_sigmaxx0_now, float * ex_sigmazz0_now, float * ex_sigmaxy0_now, float * ex_sigmaxz0_now, float * ex_sigmayz0_now,//(ny+10, nx+10, nx+10)
        float *ex_Vy0_next,  float * ex_Vx0_next, float * ex_Vz0_next, float * ex_sigmayy0_next, float *ex_sigmaxx0_next, float * ex_sigmazz0_next, float * ex_sigmaxy0_next, float * ex_sigmaxz0_next, float * ex_sigmayz0_next,//(ny+10, nx+10, nx+10) 
        float *ex_Vy0_pre,  float * ex_Vx0_pre, float * ex_Vz0_pre, float * ex_sigmayy0_pre, float *ex_sigmaxx0_pre, float * ex_sigmazz0_pre, float * ex_sigmaxy0_pre, float * ex_sigmaxz0_pre, float * ex_sigmayz0_pre,//(ny+10, nx+10, nx+10) 
        float * ex_m1_y, float * ex_m1_x,float * ex_m1_z,float * ex_m2, float * ex_m3, float * ex_m2m3,//)//(nz+10,nx+10)
	float * debug, float * gpu_kernel_time, float *source)

{	
     	hipError_t err;
	hipEvent_t start1, start2, start3, stop1, stop2, stop3;
	float elapsedTime1 = 0.0f;//time for data copy in GPU
	float elapsedTime2 = 0.0f;//time for kernel computing + overlapping
	float elapsedTime3 = 0.0f;//time for data copy out GPU

	hipSetDevice(0);
	hipEventCreate(&start1);
	hipEventCreate(&start2);
	hipEventCreate(&start3);
	hipEventCreate(&stop1);
	hipEventCreate(&stop2);
	hipEventCreate(&stop3);
	
	//config multiple GPU status
	//Acquire the number of GPU available
	int n_gpus;
#ifdef GPU_NUM
	n_gpus = GPU_NUM;
#else
	fprintf(stderr, "Please enter the number of GPU to use (between 1 and %d):\n", MAX_NUM_GPUS);
	scanf("%d", &n_gpus);
#endif

	int gpu_n;
	hipGetDeviceCount(&gpu_n);
	if(n_gpus>gpu_n){
		fprintf(stderr, "[EXIT] GPU number avalable %d less than number assigned %d\n", gpu_n, n_gpus);
		//fprintf(stderr, "Set GPU number from assigned %d to GPU number avalable %d \n", n_gpus, gpu_n);
		//n_gpus = gpu_n;
		exit(1);
	}

	//init GPU devices with P2P being enabled
	setup_cuda(n_gpus);

	// GPU inputs and outputs corresponds to CPU inputs and outputs
	float *g_ex_Vx0_now[n_gpus], *g_ex_Vx0_next[n_gpus], *g_ex_Vx0_pre[n_gpus];
	float *g_ex_Vy0_now[n_gpus], *g_ex_Vy0_next[n_gpus], *g_ex_Vy0_pre[n_gpus];
	float *g_ex_Vz0_now[n_gpus], *g_ex_Vz0_next[n_gpus], *g_ex_Vz0_pre[n_gpus];
	float *g_ex_sigmaxx0_now[n_gpus], *g_ex_sigmaxx0_next[n_gpus], *g_ex_sigmaxx0_pre[n_gpus];
	float *g_ex_sigmayy0_now[n_gpus], *g_ex_sigmayy0_next[n_gpus], *g_ex_sigmayy0_pre[n_gpus];
	float *g_ex_sigmazz0_now[n_gpus], *g_ex_sigmazz0_next[n_gpus], *g_ex_sigmazz0_pre[n_gpus];
	float *g_ex_sigmaxy0_now[n_gpus], *g_ex_sigmaxy0_next[n_gpus], *g_ex_sigmaxy0_pre[n_gpus];
	float *g_ex_sigmaxz0_now[n_gpus], *g_ex_sigmaxz0_next[n_gpus], *g_ex_sigmaxz0_pre[n_gpus];
	float *g_ex_sigmayz0_now[n_gpus], *g_ex_sigmayz0_next[n_gpus], *g_ex_sigmayz0_pre[n_gpus];

	float *g_ex_m2[n_gpus];
	float *g_ex_m3[n_gpus];
	float *g_ex_m2m3[n_gpus];
	float *g_ex_m1_x[n_gpus];
	float *g_ex_m1_z[n_gpus];
	float *g_ex_m1_y[n_gpus];
	//config index and offset

	int n1 = (NZ+10), n2 = (NX+10), n3 = (NY+10); // NZ -->NX--> NY: fast --> slow, NX, NY, NZ refers to the original index from Fortran code, and here needs to be extended by plus 10 for stencil doundary
	n3 = (n3 - 2*radius)/n_gpus + 2*radius;		
	int n_bytes_gpu = (n1*n2*n3)*sizeof(float);   //bytes length assigned for each GPU

	int start[n_gpus];				//CPU index for copy data to GPU	
	for(int i=0; i<n_gpus; i++){			//Define coordinates and offsets parameters	
		start[i] = i*(n3-2*radius) * (n1*n2);
	}



	//determin which GPU to add source (75, 100, 100) based on the number of gpus
	int gpu_id=0;
	int source_y=75;
	int n3_source = ((n3-2*radius)); 
	while((n3_source)*(gpu_id+1)<75){
		gpu_id++;
		source_y -= n3_source;
	}
	printf("%d, %d, %d\n", gpu_id, n3_source, source_y);


	//offset, bytes length, and block/grid config for internal part
	int offset_internal = radius*(n1*n2);		
	int n_bytes_gpu_internal = (n3-4*radius)*(n1*n2)*sizeof(float);
	dim3 dimGrid_internal(n1/TZ, n2/TX, (n3-4*radius)/TY);
	dim3 dimBlock(TZ, TX, TY);

	//offset, bytes length, and block/grid config for upper and bottom halo parts
	int offset_halo_up = 0;	
	int offset_halo_bt = (n3-3*radius)*(n1*n2);
	int n_bytes_gpu_halo = radius*(n1*n2)*sizeof(float);
	dim3 dimGrid_halo(n1/TZ, n2/TX, radius/TY);

	//offset and bytes length for output data back to CPU
	int offset_out = radius * n1*n2;
	int n_bytes_gpu_back = n1*n2*(n3-2*radius)*sizeof(float);

	//malloc data on each GPU
	for(int i=0; i<n_gpus; i++){
		hipSetDevice(device[i]);
		//hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		hipMalloc((void **) &g_ex_Vx0_now[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_Vx0_next[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_Vx0_pre[i], n_bytes_gpu);

		hipMalloc((void **) &g_ex_Vy0_now[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_Vy0_next[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_Vy0_pre[i], n_bytes_gpu);

		hipMalloc((void **) &g_ex_Vz0_now[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_Vz0_next[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_Vz0_pre[i], n_bytes_gpu);

		hipMalloc((void **) &g_ex_sigmaxx0_now[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmaxx0_next[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmaxx0_pre[i], n_bytes_gpu);

		hipMalloc((void **) &g_ex_sigmayy0_now[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmayy0_next[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmayy0_pre[i], n_bytes_gpu);

		hipMalloc((void **) &g_ex_sigmazz0_now[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmazz0_next[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmazz0_pre[i], n_bytes_gpu);

		hipMalloc((void **) &g_ex_sigmaxy0_now[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmaxy0_next[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmaxy0_pre[i], n_bytes_gpu);

		hipMalloc((void **) &g_ex_sigmaxz0_now[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmaxz0_next[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmaxz0_pre[i], n_bytes_gpu);

		hipMalloc((void **) &g_ex_sigmayz0_now[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmayz0_next[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_sigmayz0_pre[i], n_bytes_gpu);

		hipMalloc((void **) &g_ex_m2[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_m3[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_m2m3[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_m1_x[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_m1_y[i], n_bytes_gpu);
		hipMalloc((void **) &g_ex_m1_z[i], n_bytes_gpu);
		
	}
	
	//copy data to each GPU
	hipSetDevice(0);
	hipEventRecord(start1, 0);
	for(int i = 0; i<n_gpus; i++){

		hipSetDevice(device[i]);

		//put source on the second GPU (point with global index (75, 100, 100) )
		if(device[i]==gpu_id){
			hipMemcpyToSymbol(HIP_SYMBOL(g_source),source,TS*sizeof(float));
		}

		hipMemcpy(g_ex_Vx0_now[i],  &ex_Vx0_now[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_Vx0_next[i], &ex_Vx0_next[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_Vx0_pre[i], &ex_Vx0_pre[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);

		hipMemcpy(g_ex_Vy0_now[i],  &ex_Vy0_now[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_Vy0_next[i], &ex_Vy0_next[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_Vy0_pre[i], &ex_Vy0_pre[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);

		hipMemcpy(g_ex_Vz0_now[i],  &ex_Vz0_now[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_Vz0_next[i], &ex_Vz0_next[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_Vz0_pre[i], &ex_Vz0_pre[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);

		hipMemcpy(g_ex_sigmaxx0_now[i], &ex_sigmaxx0_now[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmaxx0_next[i], &ex_sigmaxx0_next[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmaxx0_pre[i], &ex_sigmaxx0_pre[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);

		hipMemcpy(g_ex_sigmayy0_now[i], &ex_sigmayy0_now[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmayy0_next[i], &ex_sigmayy0_next[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmayy0_pre[i], &ex_sigmayy0_pre[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);

		hipMemcpy(g_ex_sigmazz0_now[i], &ex_sigmazz0_now[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmazz0_next[i], &ex_sigmazz0_next[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmazz0_pre[i], &ex_sigmazz0_pre[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);

		hipMemcpy(g_ex_sigmaxy0_now[i], &ex_sigmaxy0_now[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmaxy0_next[i], &ex_sigmaxy0_next[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmaxy0_pre[i], &ex_sigmaxy0_pre[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);

		hipMemcpy(g_ex_sigmaxz0_now[i], &ex_sigmaxz0_now[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmaxz0_next[i], &ex_sigmaxz0_next[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmaxz0_pre[i], &ex_sigmaxz0_pre[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);

		hipMemcpy(g_ex_sigmayz0_now[i], &ex_sigmayz0_now[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmayz0_next[i], &ex_sigmayz0_next[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_sigmayz0_pre[i], &ex_sigmayz0_pre[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);

		hipMemcpy(g_ex_m2[i], &ex_m2[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_m3[i], &ex_m3[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_m2m3[i], &ex_m2m3[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_m1_x[i], &ex_m1_x[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_m1_y[i], &ex_m1_y[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
		hipMemcpy(g_ex_m1_z[i], &ex_m1_z[start[i]], n_bytes_gpu, hipMemcpyHostToDevice);
	}
	hipSetDevice(0);
	hipEventRecord(stop1, 0);
	

	//Define separate streams for overlapping communication
	hipStream_t stream_halo[n_gpus], stream_internal[n_gpus];
	for(int i=0; i<n_gpus; i++){
		hipSetDevice(device[i]);
		hipStreamCreate(&stream_halo[i]);
		hipStreamCreate(&stream_internal[i]);
	}

//-----------------------------------------------------------------------------//
//--------------------------GPU COMPUTATION------------------------------------//
//-----------------------------------------------------------------------------//

	//fprintf(stderr,"GPU Computing ... ...(NZ=%d, NX=%d, NY=%d, TZ=%d, TX=%d, TY=%d)\n", nz, nx, ny, TZ, TX, TY);
	
	hipSetDevice(0);
	hipEventRecord(start2, 0);

//Doing elastic RTM for TS time steps
	//fprintf(stderr, "[");
	for(int g_it=0; g_it<TS;g_it++){
		fprintf(stderr, "GPU at %d / %d step\n",g_it+1, TS);
	
		//Calculate the halo regions first
		for(int i=0; i<n_gpus; i++){
			hipSetDevice(i);

			//Upper Halo
			rtm_gpu_kernel<<<dimGrid_halo, dimBlock,0,stream_halo[i]>>>(ny, nz, nx,
				g_ex_Vy0_now[i] + offset_halo_up, g_ex_Vx0_now[i] + offset_halo_up, g_ex_Vz0_now[i] + offset_halo_up, g_ex_sigmayy0_now[i] + offset_halo_up, g_ex_sigmaxx0_now[i] + offset_halo_up, g_ex_sigmazz0_now[i] + offset_halo_up, g_ex_sigmaxy0_now[i] + offset_halo_up, g_ex_sigmaxz0_now[i] + offset_halo_up, g_ex_sigmayz0_now[i] + offset_halo_up,
				g_ex_Vy0_next[i] + offset_halo_up, g_ex_Vx0_next[i] + offset_halo_up, g_ex_Vz0_next[i] + offset_halo_up, g_ex_sigmayy0_next[i] + offset_halo_up, g_ex_sigmaxx0_next[i] + offset_halo_up, g_ex_sigmazz0_next[i] + offset_halo_up, g_ex_sigmaxy0_next[i] + offset_halo_up, g_ex_sigmaxz0_next[i] + offset_halo_up, g_ex_sigmayz0_next[i] + offset_halo_up,
				g_ex_Vy0_pre[i] + offset_halo_up, g_ex_Vx0_pre[i] + offset_halo_up, g_ex_Vz0_pre[i] + offset_halo_up, g_ex_sigmayy0_pre[i] + offset_halo_up, g_ex_sigmaxx0_pre[i] + offset_halo_up, g_ex_sigmazz0_pre[i] + offset_halo_up, g_ex_sigmaxy0_pre[i] + offset_halo_up, g_ex_sigmaxz0_pre[i] + offset_halo_up, g_ex_sigmayz0_pre[i] + offset_halo_up,
				g_ex_m1_y[i] + offset_halo_up, g_ex_m1_x[i] + offset_halo_up, g_ex_m1_z[i] + offset_halo_up, g_ex_m2[i] + offset_halo_up, g_ex_m3[i] + offset_halo_up, g_ex_m2m3[i] + offset_halo_up);

			//Botom Halo
			rtm_gpu_kernel<<<dimGrid_halo, dimBlock,0,stream_halo[i]>>>(ny, nz, nx,
				g_ex_Vy0_now[i] + offset_halo_bt, g_ex_Vx0_now[i] + offset_halo_bt, g_ex_Vz0_now[i] + offset_halo_bt, g_ex_sigmayy0_now[i] + offset_halo_bt, g_ex_sigmaxx0_now[i] + offset_halo_bt, g_ex_sigmazz0_now[i] + offset_halo_bt, g_ex_sigmaxy0_now[i] + offset_halo_bt, g_ex_sigmaxz0_now[i] + offset_halo_bt, g_ex_sigmayz0_now[i] + offset_halo_bt,
				g_ex_Vy0_next[i] + offset_halo_bt, g_ex_Vx0_next[i] + offset_halo_bt, g_ex_Vz0_next[i] + offset_halo_bt, g_ex_sigmayy0_next[i] + offset_halo_bt, g_ex_sigmaxx0_next[i] + offset_halo_bt, g_ex_sigmazz0_next[i] + offset_halo_bt, g_ex_sigmaxy0_next[i] + offset_halo_bt, g_ex_sigmaxz0_next[i] + offset_halo_bt, g_ex_sigmayz0_next[i] + offset_halo_bt,
				g_ex_Vy0_pre[i] + offset_halo_bt, g_ex_Vx0_pre[i] + offset_halo_bt, g_ex_Vz0_pre[i] + offset_halo_bt, g_ex_sigmayy0_pre[i] + offset_halo_bt, g_ex_sigmaxx0_pre[i] + offset_halo_bt, g_ex_sigmazz0_pre[i] + offset_halo_bt, g_ex_sigmaxy0_pre[i] + offset_halo_bt, g_ex_sigmaxz0_pre[i] + offset_halo_bt, g_ex_sigmayz0_pre[i] + offset_halo_bt,
				g_ex_m1_y[i] + offset_halo_bt, g_ex_m1_x[i] + offset_halo_bt, g_ex_m1_z[i] + offset_halo_bt, g_ex_m2[i] + offset_halo_bt, g_ex_m3[i] + offset_halo_bt, g_ex_m2m3[i] + offset_halo_bt);

			hipStreamQuery(stream_halo[i]);
		}	
	err = hipGetLastError();
	if(hipSuccess != err){
		fprintf(stderr, "Cuda error5: %s.\n", hipGetErrorString(err));
		exit(0);
	}	
	

		//Compute the internal part
		for(int i=0; i<n_gpus; i++){
			hipSetDevice(i);	

			rtm_gpu_kernel<<<dimGrid_internal, dimBlock, 0, stream_internal[i]>>>(ny, nz, nx,
				g_ex_Vy0_now[i] + offset_internal, g_ex_Vx0_now[i] + offset_internal, g_ex_Vz0_now[i] + offset_internal, g_ex_sigmayy0_now[i] + offset_internal, g_ex_sigmaxx0_now[i] + offset_internal, g_ex_sigmazz0_now[i] + offset_internal, g_ex_sigmaxy0_now[i] + offset_internal, g_ex_sigmaxz0_now[i] + offset_internal, g_ex_sigmayz0_now[i] + offset_internal,
				g_ex_Vy0_next[i] + offset_internal, g_ex_Vx0_next[i] + offset_internal, g_ex_Vz0_next[i] + offset_internal, g_ex_sigmayy0_next[i] + offset_internal, g_ex_sigmaxx0_next[i] + offset_internal, g_ex_sigmazz0_next[i] + offset_internal, g_ex_sigmaxy0_next[i] + offset_internal, g_ex_sigmaxz0_next[i] + offset_internal, g_ex_sigmayz0_next[i] + offset_internal,
			g_ex_Vy0_pre[i] + offset_internal, g_ex_Vx0_pre[i] + offset_internal, g_ex_Vz0_pre[i] + offset_internal, g_ex_sigmayy0_pre[i] + offset_internal, g_ex_sigmaxx0_pre[i] + offset_internal, g_ex_sigmazz0_pre[i] + offset_internal, g_ex_sigmaxy0_pre[i] + offset_internal, g_ex_sigmaxz0_pre[i] + offset_internal, g_ex_sigmayz0_pre[i] + offset_internal,
			g_ex_m1_y[i] + offset_internal, g_ex_m1_x[i] + offset_internal, g_ex_m1_z[i] + offset_internal, g_ex_m2[i] + offset_internal, g_ex_m3[i] + offset_internal, g_ex_m2m3[i] + offset_internal);
		}	

		//Add source to point at GPU i based on the number of GPUs
	
	err = hipGetLastError();
	if(hipSuccess != err){
		fprintf(stderr, "Cuda error5: %s.\n", hipGetErrorString(err));
		exit(0);
	}	

		hipSetDevice(gpu_id);
		dim3 dimGrid_source(n1/TZ, n2/TX, (n3-2*radius)/TY);
		add_source_kernel<<<dimGrid_source, dimBlock>>>(ny, nz, nx, source_y, 100, 100, g_it, g_ex_sigmayy0_now[gpu_id], g_ex_sigmaxx0_now[gpu_id], g_ex_sigmazz0_now[gpu_id]);
	
	err = hipGetLastError();
	if(hipSuccess != err){
		fprintf(stderr, "Cuda error5: %s.\n", hipGetErrorString(err));
		exit(0);
	}	

		//Halo updating
		if(g_it < TS-1){

			//offset for each halo parts
			int offset_down_snd = (n3-2*radius)*(n1*n2);
			int offset_down_rcv = 0;//n3-2*radius)*(n1*n2);
			int offset_up_snd   = radius*n1*n2;
			int offset_up_rcv   = (n3-radius)*n1*n2;
	
			//Send halos downwards
			for(int i=0; i<n_gpus-1; i++){
				hipMemcpyPeerAsync(g_ex_Vx0_pre[i+1]+offset_down_rcv,i+1, g_ex_Vx0_pre[i]+offset_down_snd, i, n_bytes_gpu_halo, stream_halo[i]);
				hipMemcpyPeerAsync(g_ex_Vy0_pre[i+1]+offset_down_rcv,i+1, g_ex_Vy0_pre[i]+offset_down_snd, i, n_bytes_gpu_halo, stream_halo[i]);
				hipMemcpyPeerAsync(g_ex_Vz0_pre[i+1]+offset_down_rcv,i+1, g_ex_Vz0_pre[i]+offset_down_snd, i, n_bytes_gpu_halo, stream_halo[i]);
				hipMemcpyPeerAsync(g_ex_sigmaxx0_pre[i+1]+offset_down_rcv,i+1, g_ex_sigmaxx0_pre[i]+offset_down_snd, i, n_bytes_gpu_halo, stream_halo[i]);
				hipMemcpyPeerAsync(g_ex_sigmayy0_pre[i+1]+offset_down_rcv,i+1, g_ex_sigmayy0_pre[i]+offset_down_snd, i, n_bytes_gpu_halo, stream_halo[i]);
				hipMemcpyPeerAsync(g_ex_sigmazz0_pre[i+1]+offset_down_rcv,i+1, g_ex_sigmazz0_pre[i]+offset_down_snd, i, n_bytes_gpu_halo, stream_halo[i]);
				hipMemcpyPeerAsync(g_ex_sigmaxy0_pre[i+1]+offset_down_rcv,i+1, g_ex_sigmaxy0_pre[i]+offset_down_snd, i, n_bytes_gpu_halo, stream_halo[i]);
				hipMemcpyPeerAsync(g_ex_sigmaxz0_pre[i+1]+offset_down_rcv,i+1, g_ex_sigmaxz0_pre[i]+offset_down_snd, i, n_bytes_gpu_halo, stream_halo[i]);
				hipMemcpyPeerAsync(g_ex_sigmayz0_pre[i+1]+offset_down_rcv,i+1, g_ex_sigmayz0_pre[i]+offset_down_snd, i, n_bytes_gpu_halo, stream_halo[i]);
			}		
			
			//Synchronize to avoid stalling
			for(int i=0; i<n_gpus; i++){
				hipSetDevice(i);
				hipStreamSynchronize(stream_halo[i]);
			}	

			//Send halos upwards
			for(int i=1; i<n_gpus; i++){
				hipMemcpyPeerAsync(g_ex_Vx0_pre[i-1]+offset_up_rcv,i-1, g_ex_Vx0_pre[i]+offset_up_snd, i, n_bytes_gpu_halo, stream_halo[i]);
				hipMemcpyPeerAsync(g_ex_Vy0_pre[i-1]+offset_up_rcv,i-1, g_ex_Vy0_pre[i]+offset_up_snd, i, n_bytes_gpu_halo, stream_halo[i]);
				hipMemcpyPeerAsync(g_ex_Vz0_pre[i-1]+offset_up_rcv,i-1, g_ex_Vz0_pre[i]+offset_up_snd, i, n_bytes_gpu_halo, stream_halo[i]);
				hipMemcpyPeerAsync(g_ex_sigmaxx0_pre[i-1]+offset_up_rcv,i-1, g_ex_sigmaxx0_pre[i]+offset_up_snd, i, n_bytes_gpu_halo, stream_halo[i]);
				hipMemcpyPeerAsync(g_ex_sigmayy0_pre[i-1]+offset_up_rcv,i-1, g_ex_sigmayy0_pre[i]+offset_up_snd, i, n_bytes_gpu_halo, stream_halo[i]);
				hipMemcpyPeerAsync(g_ex_sigmazz0_pre[i-1]+offset_up_rcv,i-1, g_ex_sigmazz0_pre[i]+offset_up_snd, i, n_bytes_gpu_halo, stream_halo[i]);
				hipMemcpyPeerAsync(g_ex_sigmaxz0_pre[i-1]+offset_up_rcv,i-1, g_ex_sigmaxz0_pre[i]+offset_up_snd, i, n_bytes_gpu_halo, stream_halo[i]);
				hipMemcpyPeerAsync(g_ex_sigmayz0_pre[i-1]+offset_up_rcv,i-1, g_ex_sigmayz0_pre[i]+offset_up_snd, i, n_bytes_gpu_halo, stream_halo[i]);
				hipMemcpyPeerAsync(g_ex_sigmaxy0_pre[i-1]+offset_up_rcv,i-1, g_ex_sigmaxy0_pre[i]+offset_up_snd, i, n_bytes_gpu_halo, stream_halo[i]);
			}
		}
	
		//Here we only record the kernel computing time between 0 ~ (Steps_write_back-1)
		if(g_it+1 == Steps_write_back){ 
			hipSetDevice(0);
			hipEventRecord(stop2, 0);
		}


		//Write data back to CPU and write to History Files evert Steps_write_back steps
		if((g_it+1)%Steps_write_back==0 ){	
			//Copy out data back to CPU
		
			if((g_it+1)==Steps_write_back){
				hipSetDevice(0);
				hipEventRecord(start3, 0);
			}
	
			for(int i = 0; i<n_gpus; i++){
		
				hipSetDevice(device[i]);
				hipMemcpy(&ex_Vx0_pre[start[i]]+offset_out, g_ex_Vx0_pre[i]+offset_out, n_bytes_gpu_back, hipMemcpyDeviceToHost);
				hipMemcpy(&ex_Vy0_pre[start[i]]+offset_out, g_ex_Vy0_pre[i]+offset_out, n_bytes_gpu_back, hipMemcpyDeviceToHost);
				hipMemcpy(&ex_Vz0_pre[start[i]]+offset_out, g_ex_Vz0_pre[i]+offset_out, n_bytes_gpu_back, hipMemcpyDeviceToHost);
				hipMemcpy(&ex_sigmaxx0_pre[start[i]]+offset_out, g_ex_sigmaxx0_pre[i]+offset_out, n_bytes_gpu_back, hipMemcpyDeviceToHost);
				hipMemcpy(&ex_sigmayy0_pre[start[i]]+offset_out, g_ex_sigmayy0_pre[i]+offset_out, n_bytes_gpu_back, hipMemcpyDeviceToHost);
				hipMemcpy(&ex_sigmazz0_pre[start[i]]+offset_out, g_ex_sigmazz0_pre[i]+offset_out, n_bytes_gpu_back, hipMemcpyDeviceToHost);
				hipMemcpy(&ex_sigmaxy0_pre[start[i]]+offset_out, g_ex_sigmaxy0_pre[i]+offset_out, n_bytes_gpu_back, hipMemcpyDeviceToHost);
				hipMemcpy(&ex_sigmayz0_pre[start[i]]+offset_out, g_ex_sigmayz0_pre[i]+offset_out, n_bytes_gpu_back, hipMemcpyDeviceToHost);
				hipMemcpy(&ex_sigmaxz0_pre[start[i]]+offset_out, g_ex_sigmaxz0_pre[i]+offset_out, n_bytes_gpu_back, hipMemcpyDeviceToHost);
			}


			if((g_it+1)==Steps_write_back){
				hipSetDevice(0);
				hipEventRecord(stop3, 0);
			}
	
#ifdef GPU_FILE		
			fprintf(stderr,"GPU to CPU and then write to History File at step %d\n", g_it+1);

			int ierr = srite("g_data_Vx0",ex_Vx0_pre, sizeof(float)*(nx+10)*(ny+10)*(nz+10));
			ierr = srite("g_data_Vy0",ex_Vy0_pre, sizeof(float)*(nx+10)*(ny+10)*(nz+10));
			ierr = srite("g_data_Vz0",ex_Vz0_pre, sizeof(float)*(nx+10)*(ny+10)*(nz+10));
#endif

		}	

		//Change pointer
		if(g_it < TS -1){
			float *g_tmp = NULL;
			for(int i=0; i<n_gpus; i++){
				hipSetDevice(device[i]);
	
				g_tmp = g_ex_Vx0_pre[i]; g_ex_Vx0_pre[i] = g_ex_Vx0_now[i]; g_ex_Vx0_now[i] = g_tmp;
				g_tmp = g_ex_Vx0_pre[i]; g_ex_Vx0_pre[i] = g_ex_Vx0_next[i]; g_ex_Vx0_next[i] = g_tmp; 
	
				g_tmp = g_ex_Vy0_pre[i]; g_ex_Vy0_pre[i] = g_ex_Vy0_now[i]; g_ex_Vy0_now[i] = g_tmp;
				g_tmp = g_ex_Vy0_pre[i]; g_ex_Vy0_pre[i] = g_ex_Vy0_next[i]; g_ex_Vy0_next[i] = g_tmp; 
	
				g_tmp = g_ex_Vz0_pre[i]; g_ex_Vz0_pre[i] = g_ex_Vz0_now[i]; g_ex_Vz0_now[i] = g_tmp;
				g_tmp = g_ex_Vz0_pre[i]; g_ex_Vz0_pre[i] = g_ex_Vz0_next[i]; g_ex_Vz0_next[i] = g_tmp;
	 
				g_tmp = g_ex_sigmaxx0_pre[i]; g_ex_sigmaxx0_pre[i] = g_ex_sigmaxx0_now[i]; g_ex_sigmaxx0_now[i] = g_tmp;
				g_tmp = g_ex_sigmaxx0_pre[i]; g_ex_sigmaxx0_pre[i] = g_ex_sigmaxx0_next[i]; g_ex_sigmaxx0_next[i] = g_tmp;
	 
				g_tmp = g_ex_sigmayy0_pre[i]; g_ex_sigmayy0_pre[i] = g_ex_sigmayy0_now[i]; g_ex_sigmayy0_now[i] = g_tmp;
				g_tmp = g_ex_sigmayy0_pre[i]; g_ex_sigmayy0_pre[i] = g_ex_sigmayy0_next[i]; g_ex_sigmayy0_next[i] = g_tmp;
	 
				g_tmp = g_ex_sigmazz0_pre[i]; g_ex_sigmazz0_pre[i] = g_ex_sigmazz0_now[i]; g_ex_sigmazz0_now[i] = g_tmp;
				g_tmp = g_ex_sigmazz0_pre[i]; g_ex_sigmazz0_pre[i] = g_ex_sigmazz0_next[i]; g_ex_sigmazz0_next[i] = g_tmp;
	 
				g_tmp = g_ex_sigmaxy0_pre[i]; g_ex_sigmaxy0_pre[i] = g_ex_sigmaxy0_now[i]; g_ex_sigmaxy0_now[i] = g_tmp;
				g_tmp = g_ex_sigmaxy0_pre[i]; g_ex_sigmaxy0_pre[i] = g_ex_sigmaxy0_next[i]; g_ex_sigmaxy0_next[i] = g_tmp; 
	
				g_tmp = g_ex_sigmaxz0_pre[i]; g_ex_sigmaxz0_pre[i] = g_ex_sigmaxz0_now[i]; g_ex_sigmaxz0_now[i] = g_tmp;
				g_tmp = g_ex_sigmaxz0_pre[i]; g_ex_sigmaxz0_pre[i] = g_ex_sigmaxz0_next[i]; g_ex_sigmaxz0_next[i] = g_tmp;
	 
				g_tmp = g_ex_sigmayz0_pre[i]; g_ex_sigmayz0_pre[i] = g_ex_sigmayz0_now[i]; g_ex_sigmayz0_now[i] = g_tmp;
				g_tmp = g_ex_sigmayz0_pre[i]; g_ex_sigmayz0_pre[i] = g_ex_sigmayz0_next[i]; g_ex_sigmayz0_next[i] = g_tmp; 
	
			}
		}
		
		//if(g_it%(TS/40)==0) fprintf(stderr, "#");
	}
	//fprintf(stderr, "]\n");	

	hipEventSynchronize(stop1);
	hipEventSynchronize(stop2);
	hipEventSynchronize(stop3);

	hipEventElapsedTime(&elapsedTime1, start1, stop1);
	hipEventElapsedTime(&elapsedTime2, start2, stop2);
	hipEventElapsedTime(&elapsedTime3, start3, stop3);


	gpu_kernel_time[0] = (float)(elapsedTime1/1000.);
	gpu_kernel_time[1] = (float)(elapsedTime2/1000.);
	gpu_kernel_time[2] = (float)(elapsedTime3/1000.);
	
	hipEventDestroy(start1);
	hipEventDestroy(start2);
	hipEventDestroy(start3);
	hipEventDestroy(stop1);
	hipEventDestroy(stop2);
	hipEventDestroy(stop3);
	
	for(int i=0; i<n_gpus; i++){
		hipSetDevice(i);

		hipFree(g_ex_Vx0_now[i]);
		hipFree(g_ex_Vz0_now[i]);
		hipFree(g_ex_Vy0_now[i]);
		hipFree(g_ex_sigmaxx0_now[i]);
		hipFree(g_ex_sigmazz0_now[i]);
		hipFree(g_ex_sigmayy0_now[i]);
		hipFree(g_ex_sigmaxy0_now[i]);
		hipFree(g_ex_sigmaxz0_now[i]);
		hipFree(g_ex_sigmayz0_now[i]);
		
		//Time step +2
		hipFree(g_ex_Vx0_next[i]);
		hipFree(g_ex_Vz0_next[i]);
		hipFree(g_ex_Vy0_next[i]);
		hipFree(g_ex_sigmaxx0_next[i]);
		hipFree(g_ex_sigmazz0_next[i]);
		hipFree(g_ex_sigmayy0_next[i]);
		hipFree(g_ex_sigmaxy0_next[i]);
		hipFree(g_ex_sigmaxz0_next[i]);
		hipFree(g_ex_sigmayz0_next[i]);
	
	
		//time step 0 and output
		hipFree(g_ex_Vx0_pre[i]);
		hipFree(g_ex_Vz0_pre[i]);
		hipFree(g_ex_Vy0_pre[i]);
		hipFree(g_ex_sigmaxx0_pre[i]);
		hipFree(g_ex_sigmazz0_pre[i]);
		hipFree(g_ex_sigmayy0_pre[i]);
		hipFree(g_ex_sigmaxy0_pre[i]);
		hipFree(g_ex_sigmaxz0_pre[i]);
		hipFree(g_ex_sigmayz0_pre[i]);
	   
		//expaned arrays to store different Operators 
		hipFree(g_ex_m2[i]);
		hipFree(g_ex_m3[i]);
		hipFree(g_ex_m2m3[i]);
		hipFree(g_ex_m1_x[i]);
		hipFree(g_ex_m1_y[i]);
		hipFree(g_ex_m1_z[i]);
		}
}

